#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright: 2010-2014, iMinds-Vision Lab, University of Antwerp
                2014, CWI, Amsterdam

Contact: astra@uantwerpen.be
Website: http://sf.net/projects/astra-toolbox

This file is part of the ASTRA Toolbox.


The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "cone_fp.h"
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneProjTexture;

namespace astraCUDA3d {

static const unsigned int g_volBlockZ = 16;

static const unsigned int g_anglesPerBlock = 64;
static const unsigned int g_volBlockX = 32;
static const unsigned int g_volBlockY = 16;

static const unsigned g_MaxAngles = 1024;

__constant__ float gC_Cux[g_MaxAngles];
__constant__ float gC_Cuy[g_MaxAngles];
__constant__ float gC_Cuz[g_MaxAngles];
__constant__ float gC_Cuc[g_MaxAngles];
__constant__ float gC_Cvx[g_MaxAngles];
__constant__ float gC_Cvy[g_MaxAngles];
__constant__ float gC_Cvz[g_MaxAngles];
__constant__ float gC_Cvc[g_MaxAngles];
__constant__ float gC_Cdx[g_MaxAngles];
__constant__ float gC_Cdy[g_MaxAngles];
__constant__ float gC_Cdz[g_MaxAngles];
__constant__ float gC_Cdc[g_MaxAngles];


bool bindProjDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneProjTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneProjTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneProjTexture.filterMode = hipFilterModeLinear;
	gT_coneProjTexture.normalized = false;

	hipBindTextureToArray(gT_coneProjTexture, array, channelDesc);

	// TODO: error value?

	return true;
}


__global__ void dev_cone_BP(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, const SDimensions3D dims)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f;
	float fY = Y - 0.5f*dims.iVolY + 0.5f;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f;

	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			const float fCux = gC_Cux[angle];
			const float fCuy = gC_Cuy[angle];
			const float fCuz = gC_Cuz[angle];
			const float fCuc = gC_Cuc[angle];
			const float fCvx = gC_Cvx[angle];
			const float fCvy = gC_Cvy[angle];
			const float fCvz = gC_Cvz[angle];
			const float fCvc = gC_Cvc[angle];
			const float fCdx = gC_Cdx[angle];
			const float fCdy = gC_Cdy[angle];
			const float fCdz = gC_Cdz[angle];
			const float fCdc = gC_Cdc[angle];

			const float fUNum = fCuc + fX * fCux + fY * fCuy + fZ * fCuz;
			const float fVNum = fCvc + fX * fCvx + fY * fCvy + fZ * fCvz;
			const float fDen = fCdc + fX * fCdx + fY * fCdy + fZ * fCdz;

			const float fU = fUNum / fDen;
			const float fV = fVNum / fDen;

			fVal += tex3D(gT_coneProjTexture, fU, fAngle, fV);

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal;
	}

}

// supersampling version
__global__ void dev_cone_BP_SS(void* D_volData, unsigned int volPitch, int startAngle, int angleOffset, const SDimensions3D dims)
{
	float* volData = (float*)D_volData;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles - angleOffset)
		endAngle = dims.iProjAngles - angleOffset;

	// threadIdx: x = rel x
	//            y = rel y

	// blockIdx:  x = x + y
    //            y = z


	// TO TRY: precompute part of detector intersection formulas in shared mem?
	// TO TRY: inner loop over z, gather ray values in shared mem

	const int X = blockIdx.x % ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockX + threadIdx.x;
	const int Y = blockIdx.x / ((dims.iVolX+g_volBlockX-1)/g_volBlockX) * g_volBlockY + threadIdx.y;

	if (X >= dims.iVolX)
		return;
	if (Y >= dims.iVolY)
		return;

	const int startZ = blockIdx.y * g_volBlockZ;
	int endZ = startZ + g_volBlockZ;
	if (endZ > dims.iVolZ)
		endZ = dims.iVolZ;

	float fX = X - 0.5f*dims.iVolX + 0.5f - 0.5f + 0.5f/dims.iRaysPerVoxelDim;
	float fY = Y - 0.5f*dims.iVolY + 0.5f - 0.5f + 0.5f/dims.iRaysPerVoxelDim;
	float fZ = startZ - 0.5f*dims.iVolZ + 0.5f - 0.5f + 0.5f/dims.iRaysPerVoxelDim;
	const float fSubStep = 1.0f/dims.iRaysPerVoxelDim;

	for (int Z = startZ; Z < endZ; ++Z, fZ += 1.0f)
	{

		float fVal = 0.0f;
		float fAngle = startAngle + angleOffset + 0.5f;

		for (int angle = startAngle; angle < endAngle; ++angle, fAngle += 1.0f)
		{

			const float fCux = gC_Cux[angle];
			const float fCuy = gC_Cuy[angle];
			const float fCuz = gC_Cuz[angle];
			const float fCuc = gC_Cuc[angle];
			const float fCvx = gC_Cvx[angle];
			const float fCvy = gC_Cvy[angle];
			const float fCvz = gC_Cvz[angle];
			const float fCvc = gC_Cvc[angle];
			const float fCdx = gC_Cdx[angle];
			const float fCdy = gC_Cdy[angle];
			const float fCdz = gC_Cdz[angle];
			const float fCdc = gC_Cdc[angle];

			float fXs = fX;
			for (int iSubX = 0; iSubX < dims.iRaysPerVoxelDim; ++iSubX) {
			float fYs = fY;
			for (int iSubY = 0; iSubY < dims.iRaysPerVoxelDim; ++iSubY) {
			float fZs = fZ;
			for (int iSubZ = 0; iSubZ < dims.iRaysPerVoxelDim; ++iSubZ) {

				const float fUNum = fCuc + fXs * fCux + fYs * fCuy + fZs * fCuz;
				const float fVNum = fCvc + fXs * fCvx + fYs * fCvy + fZs * fCvz;
				const float fDen = fCdc + fXs * fCdx + fYs * fCdy + fZs * fCdz;

				const float fU = fUNum / fDen;
				const float fV = fVNum / fDen;

				fVal += tex3D(gT_coneProjTexture, fU, fAngle, fV);

				fZs += fSubStep;
			}
			fYs += fSubStep;
			}
			fXs += fSubStep;
			}

		}

		volData[(Z*dims.iVolY+Y)*volPitch+X] += fVal / (dims.iRaysPerVoxelDim*dims.iRaysPerVoxelDim*dims.iRaysPerVoxelDim);
	}

}


bool ConeBP_Array(hipPitchedPtr D_volumeData,
                  hipArray *D_projArray,
                  const SDimensions3D& dims, const SConeProjection* angles)
{
	bindProjDataTexture(D_projArray);

	for (unsigned int th = 0; th < dims.iProjAngles; th += g_MaxAngles) {
		unsigned int angleCount = g_MaxAngles;
		if (th + angleCount > dims.iProjAngles)
			angleCount = dims.iProjAngles - th;

		// transfer angles to constant memory
		float* tmp = new float[angleCount];


		// NB: We increment angles at the end of the loop body.


		// TODO: Use functions from dims3d.cu for this:

#define TRANSFER_TO_CONSTANT(expr,name) do { for (unsigned int i = 0; i < angleCount; ++i) tmp[i] = (expr) ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, angleCount*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

		TRANSFER_TO_CONSTANT( (angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetVY - (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetVZ , Cux );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetVZ -(angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetVX , Cuy );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetVX - (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetVY , Cuz );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSY*angles[i].fDetVZ - angles[i].fDetSZ*angles[i].fDetVY)*angles[i].fSrcX - (angles[i].fDetSX*angles[i].fDetVZ - angles[i].fDetSZ*angles[i].fDetVX)*angles[i].fSrcY + (angles[i].fDetSX*angles[i].fDetVY - angles[i].fDetSY*angles[i].fDetVX)*angles[i].fSrcZ , Cuc );

		TRANSFER_TO_CONSTANT( (angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetUZ-(angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetUY, Cvx );
		TRANSFER_TO_CONSTANT( (angles[i].fDetSZ - angles[i].fSrcZ)*angles[i].fDetUX - (angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetUZ , Cvy );
		TRANSFER_TO_CONSTANT((angles[i].fDetSX - angles[i].fSrcX)*angles[i].fDetUY-(angles[i].fDetSY - angles[i].fSrcY)*angles[i].fDetUX , Cvz );
		TRANSFER_TO_CONSTANT( -(angles[i].fDetSY*angles[i].fDetUZ - angles[i].fDetSZ*angles[i].fDetUY)*angles[i].fSrcX + (angles[i].fDetSX*angles[i].fDetUZ - angles[i].fDetSZ*angles[i].fDetUX)*angles[i].fSrcY - (angles[i].fDetSX*angles[i].fDetUY - angles[i].fDetSY*angles[i].fDetUX)*angles[i].fSrcZ , Cvc );

		TRANSFER_TO_CONSTANT( angles[i].fDetUY*angles[i].fDetVZ - angles[i].fDetUZ*angles[i].fDetVY , Cdx );
		TRANSFER_TO_CONSTANT( angles[i].fDetUZ*angles[i].fDetVX - angles[i].fDetUX*angles[i].fDetVZ , Cdy );
		TRANSFER_TO_CONSTANT( angles[i].fDetUX*angles[i].fDetVY - angles[i].fDetUY*angles[i].fDetVX , Cdz );
		TRANSFER_TO_CONSTANT( -angles[i].fSrcX * (angles[i].fDetUY*angles[i].fDetVZ - angles[i].fDetUZ*angles[i].fDetVY) - angles[i].fSrcY * (angles[i].fDetUZ*angles[i].fDetVX - angles[i].fDetUX*angles[i].fDetVZ) - angles[i].fSrcZ * (angles[i].fDetUX*angles[i].fDetVY - angles[i].fDetUY*angles[i].fDetVX) , Cdc );

#undef TRANSFER_TO_CONSTANT

		delete[] tmp;

		dim3 dimBlock(g_volBlockX, g_volBlockY);

		dim3 dimGrid(((dims.iVolX+g_volBlockX-1)/g_volBlockX)*((dims.iVolY+g_volBlockY-1)/g_volBlockY), (dims.iVolZ+g_volBlockZ-1)/g_volBlockZ);

		// timeval t;
		// tic(t);

		for (unsigned int i = 0; i < angleCount; i += g_anglesPerBlock) {
		// printf("Calling BP: %d, %dx%d, %dx%d to %p\n", i, dimBlock.x, dimBlock.y, dimGrid.x, dimGrid.y, (void*)D_volumeData.ptr); 
			if (dims.iRaysPerVoxelDim == 1)
				dev_cone_BP<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims);
			else
				dev_cone_BP_SS<<<dimGrid, dimBlock>>>(D_volumeData.ptr, D_volumeData.pitch/sizeof(float), i, th, dims);
		}

		cudaTextForceKernelsCompletion();

		angles = angles + angleCount;
		// printf("%f\n", toc(t));

	}


	return true;
}

bool ConeBP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles)
{
	// transfer projections to array

	hipArray* cuArray = allocateProjectionArray(dims);
	transferProjectionsToArray(D_projData, cuArray, dims);

	bool ret = ConeBP_Array(D_volumeData, cuArray, dims, angles);

	hipFreeArray(cuArray);

	return ret;
}


}

#ifdef STANDALONE
int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 180;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
#if 0
		if (i == 128) {
			for (unsigned int j = 0; j < 256*256; ++j)
				slice[j] = 0.0f;
		}
#endif 
	}


	SConeProjection angle[180];
	angle[0].fSrcX = -1536;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 0;

	angle[0].fDetSX = 512;
	angle[0].fDetSY = -256;
	angle[0].fDetSZ = -256;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = 1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 180; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*2*M_PI/180);
		ROTATE0(DetS, i, i*2*M_PI/180);
		ROTATE0(DetU, i, i*2*M_PI/180);
		ROTATE0(DetV, i, i*2*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);
#if 0
	float* bufs = new float[180*512];

	for (int i = 0; i < 512; ++i) {
		hipMemcpy(bufs, ((float*)projData.ptr)+180*512*i, 180*512*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

		char fname[20];
		sprintf(fname, "sino%03d.png", i);
		saveImage(fname, 180, 512, bufs);
	}

	float* bufp = new float[512*512];

	for (int i = 0; i < 180; ++i) {
		for (int j = 0; j < 512; ++j) {
			hipMemcpy(bufp+512*j, ((float*)projData.ptr)+180*512*j+512*i, 512*sizeof(float), hipMemcpyDeviceToHost);
		}

		char fname[20];
		sprintf(fname, "proj%03d.png", i);
		saveImage(fname, 512, 512, bufp);
	}
#endif		
	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 0.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipMemcpy3D(&p);
	}

	astraCUDA3d::ConeBP(volData, projData, dims, angle);
#if 0
	float* buf = new float[256*256];

	for (int i = 0; i < 256; ++i) {
		hipMemcpy(buf, ((float*)volData.ptr)+256*256*i, 256*256*sizeof(float), hipMemcpyDeviceToHost);

		printf("%d %d %d\n", volData.pitch, volData.xsize, volData.ysize);

		char fname[20];
		sprintf(fname, "vol%03d.png", i);
		saveImage(fname, 256, 256, buf);
	}
#endif

}
#endif
