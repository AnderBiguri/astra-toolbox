#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include "util3d.h"
#include "../2d/util.h"

namespace astraCUDA3d {


hipPitchedPtr allocateVolumeData(const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData;

	hipError_t err = hipMalloc3D(&volData, extentV);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		fprintf(stderr, "Failed to allocate %dx%dx%d GPU buffer\n", dims.iVolX, dims.iVolY, dims.iVolZ);
		volData.ptr = 0;
		// TODO: return 0 somehow?
	}

	return volData;
}
hipPitchedPtr allocateProjectionData(const SDimensions3D& dims)
{
	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjAngles;
	extentP.depth = dims.iProjV;

	hipPitchedPtr projData;

	hipError_t err = hipMalloc3D(&projData, extentP);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		fprintf(stderr, "Failed to allocate %dx%dx%d GPU buffer\n", dims.iProjU, dims.iProjAngles, dims.iProjV);
		projData.ptr = 0;
		// TODO: return 0 somehow?
	}

	return projData;
}
bool zeroVolumeData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;
	hipError_t err;

	for (unsigned int z = 0; z < dims.iVolZ; ++z) {
		err = hipMemset2D(t, D_data.pitch, 0, dims.iVolX*sizeof(float), dims.iVolY);
		ASTRA_CUDA_ASSERT(err);
		t += D_data.pitch * dims.iVolY;
	}
	return true;
}
bool zeroProjectionData(hipPitchedPtr& D_data, const SDimensions3D& dims)
{
	char* t = (char*)D_data.ptr;
	hipError_t err;

	for (unsigned int z = 0; z < dims.iProjV; ++z) {
		err = hipMemset2D(t, D_data.pitch, 0, dims.iProjU*sizeof(float), dims.iProjAngles);
		ASTRA_CUDA_ASSERT(err);
		t += D_data.pitch * dims.iProjAngles;
	}

	return true;
}
bool copyVolumeToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool copyProjectionsToDevice(const float* data, hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = (void*)data; // const cast away
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = ptr;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_data;
	p.extent = extentV;
	p.kind = hipMemcpyHostToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool copyVolumeFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iVolX;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iVolX*sizeof(float);
	ptr.ysize = dims.iVolY;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}
bool copyProjectionsFromDevice(float* data, const hipPitchedPtr& D_data, const SDimensions3D& dims, unsigned int pitch)
{
	if (!pitch)
		pitch = dims.iProjU;

	hipPitchedPtr ptr;
	ptr.ptr = data;
	ptr.pitch = pitch*sizeof(float);
	ptr.xsize = dims.iProjU*sizeof(float);
	ptr.ysize = dims.iProjAngles;

	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_data;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = ptr;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToHost;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}

bool duplicateVolumeData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}
bool duplicateProjectionData(hipPitchedPtr& D_dst, const hipPitchedPtr& D_src, const SDimensions3D& dims)
{
	hipExtent extentV;
	extentV.width = dims.iProjU*sizeof(float);
	extentV.height = dims.iProjAngles;
	extentV.depth = dims.iProjV;

	hipPos zp = { 0, 0, 0 };

	hipMemcpy3DParms p;
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_src;
	p.dstArray = 0;
	p.dstPos = zp;
	p.dstPtr = D_dst;
	p.extent = extentV;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err;
	err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	return err == hipSuccess;
}



// TODO: Consider using a single array of size max(proj,volume) (per dim)
//       instead of allocating a new one each time

// TODO: Figure out a faster way of zeroing the padding?

hipArray* allocateVolumeArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iVolX+2;
	extentA.height = dims.iVolY+2;
	extentA.depth = dims.iVolZ+2;
	hipError_t err = hipMalloc3DArray(&cuArray, &channelDesc, extentA);
	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		fprintf(stderr, "Failed to allocate %dx%dx%d GPU array\n", dims.iVolX, dims.iVolY, dims.iVolZ);
		return 0;
	}

	zeroVolumeArray(cuArray, dims);

	return cuArray;
}
hipArray* allocateProjectionArray(const SDimensions3D& dims)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray;
	hipExtent extentA;
	extentA.width = dims.iProjU+2;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV+2;
	hipError_t err = hipMalloc3DArray(&cuArray, &channelDesc, extentA);

	if (err != hipSuccess) {
		astraCUDA::reportCudaError(err);
		fprintf(stderr, "Failed to allocate %dx%dx%d GPU array\n", dims.iProjU, dims.iProjAngles, dims.iProjV);
		return 0;
	}

	zeroProjectionArray(cuArray, dims);

	return cuArray;
}
bool zeroVolumeArray(hipArray* array, const SDimensions3D& dims)
{
	hipPitchedPtr zeroBuf;
	hipExtent extentS;
	extentS.width = sizeof(float)*(dims.iVolX+2);
	extentS.height = dims.iVolY+2;
	extentS.depth = 1;

	hipExtent extentA;
	extentA.width = dims.iVolX+2;
	extentA.height = dims.iVolY+2;
	extentA.depth = 1;



	hipError_t err;
	err = hipMalloc3D(&zeroBuf, extentS);
	ASTRA_CUDA_ASSERT(err);
	err = hipMemset2D(zeroBuf.ptr, zeroBuf.pitch, 0, sizeof(float)*(dims.iVolX+2), dims.iVolY+2);
	ASTRA_CUDA_ASSERT(err);
	
	// zero array
	for (unsigned int i = 0; i < dims.iVolZ+2; ++i) {
		hipMemcpy3DParms p;
		hipPos zp = {0, 0, 0};
		hipPos dp = {0, 0, i};
		p.srcArray = 0;
		p.srcPos = zp;
		p.srcPtr = zeroBuf;
		p.dstArray = array;
		p.dstPtr.ptr = 0;
		p.dstPtr.pitch = 0;
		p.dstPtr.xsize = 0;
		p.dstPtr.ysize = 0;
		p.dstPos = dp;
		p.extent = extentA;
		p.kind = hipMemcpyDeviceToDevice;

		err = hipMemcpy3D(&p);
		ASTRA_CUDA_ASSERT(err);
	}
	hipFree(zeroBuf.ptr);

	// TODO: check errors

	return true;
}
bool zeroProjectionArray(hipArray* array, const SDimensions3D& dims)
{
	hipPitchedPtr zeroBuf;
	hipExtent extentS;
	extentS.width = sizeof(float)*(dims.iProjU+2);
	extentS.height = dims.iProjAngles;
	extentS.depth = 1;
	hipExtent extentA;
	extentA.width = dims.iProjU+2;
	extentA.height = dims.iProjAngles;
	extentA.depth = 1;


	hipError_t err;
	err = hipMalloc3D(&zeroBuf, extentS);
	ASTRA_CUDA_ASSERT(err);
	err = hipMemset2D(zeroBuf.ptr, zeroBuf.pitch, 0, sizeof(float)*(dims.iProjU+2), dims.iProjAngles);
	ASTRA_CUDA_ASSERT(err);

	for (unsigned int i = 0; i < dims.iProjV+2; ++i) {
		hipMemcpy3DParms p;
		hipPos zp = {0, 0, 0};
		hipPos dp = {0, 0, i};
		p.srcArray = 0;
		p.srcPos = zp;
		p.srcPtr = zeroBuf;
		p.dstArray = array;
		p.dstPtr.ptr = 0;
		p.dstPtr.pitch = 0;
		p.dstPtr.xsize = 0;
		p.dstPtr.ysize = 0;
		p.dstPos = dp;
		p.extent = extentA;
		p.kind = hipMemcpyDeviceToDevice;

		err = hipMemcpy3D(&p);
		ASTRA_CUDA_ASSERT(err);
	}
	hipFree(zeroBuf.ptr);

	// TODO: check errors
	return true;
}


bool transferVolumeToArray(hipPitchedPtr D_volumeData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iVolX;
	extentA.height = dims.iVolY;
	extentA.depth = dims.iVolZ;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	hipPos dp = {1, 1, 1};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_volumeData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = dp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);
	// TODO: check errors

	return true;
}
bool transferProjectionsToArray(hipPitchedPtr D_projData, hipArray* array, const SDimensions3D& dims)
{
	hipExtent extentA;
	extentA.width = dims.iProjU;
	extentA.height = dims.iProjAngles;
	extentA.depth = dims.iProjV;

	hipMemcpy3DParms p;
	hipPos zp = {0, 0, 0};
	hipPos dp = {1, 0, 1};
	p.srcArray = 0;
	p.srcPos = zp;
	p.srcPtr = D_projData;
	p.dstArray = array;
	p.dstPtr.ptr = 0;
	p.dstPtr.pitch = 0;
	p.dstPtr.xsize = 0;
	p.dstPtr.ysize = 0;
	p.dstPos = dp;
	p.extent = extentA;
	p.kind = hipMemcpyDeviceToDevice;

	hipError_t err = hipMemcpy3D(&p);
	ASTRA_CUDA_ASSERT(err);

	// TODO: check errors

	return true;
}


float dotProduct3D(hipPitchedPtr data, unsigned int x, unsigned int y,
                   unsigned int z)
{
	return astraCUDA::dotProduct2D((float*)data.ptr, data.pitch/sizeof(float), x, y*z, 0, 0);
}


bool cudaTextForceKernelsCompletion()
{
	hipError_t returnedCudaError = hipDeviceSynchronize();

	if(returnedCudaError != hipSuccess) {
		fprintf(stderr, "Failed to force completion of cuda kernels: %d: %s.\n", returnedCudaError, hipGetErrorString(returnedCudaError));
		return false;
	}

	return true;
}

int calcNextPowerOfTwo(int _iValue)
{
	int iOutput = 1;
	while(iOutput < _iValue)
		iOutput *= 2;
	return iOutput;
}

}
