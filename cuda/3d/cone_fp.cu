#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>
#include <list>

#include <hip/hip_runtime.h>
#include "util3d.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#include "dims3d.h"

typedef texture<float, 3, hipReadModeElementType> texture3D;

static texture3D gT_coneVolumeTexture;

namespace astraCUDA3d {

static const unsigned int g_anglesPerBlock = 4;

// thickness of the slices we're splitting the volume up into
static const unsigned int g_blockSlices = 64;
static const unsigned int g_detBlockU = 32;
static const unsigned int g_detBlockV = 32;

static const unsigned g_MaxAngles = 1024;
__constant__ float gC_SrcX[g_MaxAngles];
__constant__ float gC_SrcY[g_MaxAngles];
__constant__ float gC_SrcZ[g_MaxAngles];
__constant__ float gC_DetSX[g_MaxAngles];
__constant__ float gC_DetSY[g_MaxAngles];
__constant__ float gC_DetSZ[g_MaxAngles];
__constant__ float gC_DetUX[g_MaxAngles];
__constant__ float gC_DetUY[g_MaxAngles];
__constant__ float gC_DetUZ[g_MaxAngles];
__constant__ float gC_DetVX[g_MaxAngles];
__constant__ float gC_DetVY[g_MaxAngles];
__constant__ float gC_DetVZ[g_MaxAngles];


bool bindVolumeDataTexture(const hipArray* array)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_coneVolumeTexture.addressMode[0] = hipAddressModeBorder;
	gT_coneVolumeTexture.addressMode[1] = hipAddressModeBorder;
	gT_coneVolumeTexture.addressMode[2] = hipAddressModeBorder;
	gT_coneVolumeTexture.filterMode = hipFilterModeLinear;
	gT_coneVolumeTexture.normalized = false;

	hipBindTextureToArray(gT_coneVolumeTexture, array, channelDesc);

	// TODO: error value?

	return true;
}

	// threadIdx: x = ??? detector  (u?)
	//            y = relative angle

	// blockIdx:  x = ??? detector  (u+v?)
    //            y = angle block


#define CONE_FP_BODY(c0,c1,c2) \
	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;                                          \
	if (angle >= endAngle)                                                                                         \
		return;                                                                                                    \
                                                                                                                   \
	const float fSrcX = gC_SrcX[angle];                                                                            \
	const float fSrcY = gC_SrcY[angle];                                                                            \
	const float fSrcZ = gC_SrcZ[angle];                                                                            \
	const float fDetUX = gC_DetUX[angle];                                                                          \
	const float fDetUY = gC_DetUY[angle];                                                                          \
	const float fDetUZ = gC_DetUZ[angle];                                                                          \
	const float fDetVX = gC_DetVX[angle];                                                                          \
	const float fDetVY = gC_DetVY[angle];                                                                          \
	const float fDetVZ = gC_DetVZ[angle];                                                                          \
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;                                          \
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;                                          \
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;                                          \
                                                                                                                   \
	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;      \
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;               \
	int endDetectorV = startDetectorV + g_detBlockV;                                                               \
	if (endDetectorV > dims.iProjV)                                                                                \
		endDetectorV = dims.iProjV;                                                                                \
                                                                                                                   \
	int endSlice = startSlice + g_blockSlices;                                                                     \
	if (endSlice > dims.iVol##c0)                                                                                     \
		endSlice = dims.iVol##c0;                                                                                    \
                                                                                                                   \
	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)                                    \
	{                                                                                                              \
		/* Trace ray from Src to (detectorU,detectorV) from */                                                     \
		/* X = startSlice to X = endSlice                   */                                                     \
                                                                                                                   \
		const float fDetX = fDetSX + detectorU*fDetUX + detectorV*fDetVX;                                          \
		const float fDetY = fDetSY + detectorU*fDetUY + detectorV*fDetVY;                                          \
		const float fDetZ = fDetSZ + detectorU*fDetUZ + detectorV*fDetVZ;                                          \
                                                                                                                   \
		/*        (x)   ( 1)       ( 0) */                                                                         \
		/* ray:   (y) = (ay) * x + (by) */                                                                         \
		/*        (z)   (az)       (bz) */                                                                         \
                                                                                                                   \
		const float a##c1 = (fSrc##c1 - fDet##c1) / (fSrc##c0 - fDet##c0);                                         \
		const float a##c2 = (fSrc##c2 - fDet##c2) / (fSrc##c0 - fDet##c0);                                         \
		const float b##c1 = fSrc##c1 - a##c1 * fSrc##c0;                                                           \
		const float b##c2 = fSrc##c2 - a##c2 * fSrc##c0;                                                           \
                                                                                                                   \
		const float fDistCorr = sqrt(a##c1*a##c1+a##c2*a##c2+1.0f) * fOutputScale;                                 \
                                                                                                                   \
		float fVal = 0.0f;                                                                                         \
                                                                                                                   \
		float f##c0 = startSlice + 0.5f;                                                                           \
		float f##c1 = a##c1 * (startSlice - 0.5f*dims.iVol##c0 + 0.5f) + b##c1 + 0.5f*dims.iVol##c1 - 0.5f + 0.5f; \
		float f##c2 = a##c2 * (startSlice - 0.5f*dims.iVol##c0 + 0.5f) + b##c2 + 0.5f*dims.iVol##c2 - 0.5f + 0.5f; \
                                                                                                                   \
		for (int s = startSlice; s < endSlice; ++s)                                                                \
		{                                                                                                          \
			fVal += tex3D(gT_coneVolumeTexture, fX, fY, fZ);                                                       \
			f##c0 += 1.0f;                                                                                         \
			f##c1 += a##c1;                                                                                        \
			f##c2 += a##c2;                                                                                        \
		}                                                                                                          \
                                                                                                                   \
		fVal *= fDistCorr;                                                                                         \
                                                                                                                   \
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fVal;                                \
	}

#define CONE_FP_SS_BODY(c0,c1,c2) \
	int angle = startAngle + blockIdx.y * g_anglesPerBlock + threadIdx.y;                                          \
	if (angle >= endAngle)                                                                                         \
		return;                                                                                                    \
                                                                                                                   \
	const float fSrcX = gC_SrcX[angle];                                                                            \
	const float fSrcY = gC_SrcY[angle];                                                                            \
	const float fSrcZ = gC_SrcZ[angle];                                                                            \
	const float fDetUX = gC_DetUX[angle];                                                                          \
	const float fDetUY = gC_DetUY[angle];                                                                          \
	const float fDetUZ = gC_DetUZ[angle];                                                                          \
	const float fDetVX = gC_DetVX[angle];                                                                          \
	const float fDetVY = gC_DetVY[angle];                                                                          \
	const float fDetVZ = gC_DetVZ[angle];                                                                          \
	const float fDetSX = gC_DetSX[angle] + 0.5f * fDetUX + 0.5f * fDetVX;                                          \
	const float fDetSY = gC_DetSY[angle] + 0.5f * fDetUY + 0.5f * fDetVY;                                          \
	const float fDetSZ = gC_DetSZ[angle] + 0.5f * fDetUZ + 0.5f * fDetVZ;                                          \
                                                                                                                   \
	const int detectorU = (blockIdx.x%((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockU + threadIdx.x;      \
	const int startDetectorV = (blockIdx.x/((dims.iProjU+g_detBlockU-1)/g_detBlockU)) * g_detBlockV;               \
	int endDetectorV = startDetectorV + g_detBlockV;                                                               \
	if (endDetectorV > dims.iProjV)                                                                                \
		endDetectorV = dims.iProjV;                                                                                \
                                                                                                                   \
	int endSlice = startSlice + g_blockSlices;                                                                     \
	if (endSlice > dims.iVolX)                                                                                     \
		endSlice = dims.iVolX;                                                                                     \
                                                                                                                   \
	const float fSubStep = 1.0f/dims.iRaysPerDetDim;                                                               \
                                                                                                                   \
	for (int detectorV = startDetectorV; detectorV < endDetectorV; ++detectorV)                                    \
	{                                                                                                              \
		/* Trace ray from Src to (detectorU,detectorV) from */                                                     \
		/* X = startSlice to X = endSlice                   */                                                     \
                                                                                                                   \
		float fV = 0.0f;                                                                                           \
                                                                                                                   \
		float fdU = detectorU - 0.5f + 0.5f*fSubStep;                                                              \
		for (int iSubU = 0; iSubU < dims.iRaysPerDetDim; ++iSubU, fdU+=fSubStep) {                                 \
		float fdV = detectorV - 0.5f + 0.5f*fSubStep;                                                              \
		for (int iSubV = 0; iSubV < dims.iRaysPerDetDim; ++iSubV, fdV+=fSubStep) {                                 \
                                                                                                                   \
		const float fDetX = fDetSX + fdU*fDetUX + fdV*fDetVX;                                                      \
		const float fDetY = fDetSY + fdU*fDetUY + fdV*fDetVY;                                                      \
		const float fDetZ = fDetSZ + fdU*fDetUZ + fdV*fDetVZ;                                                      \
                                                                                                                   \
		/*        (x)   ( 1)       ( 0) */                                                                         \
		/* ray:   (y) = (ay) * x + (by) */                                                                         \
		/*        (z)   (az)       (bz) */                                                                         \
                                                                                                                   \
		const float a##c1 = (fSrc##c1 - fDet##c1) / (fSrc##c0 - fDet##c0);                                         \
		const float a##c2 = (fSrc##c2 - fDet##c2) / (fSrc##c0 - fDet##c0);                                         \
		const float b##c1 = fSrc##c1 - a##c1 * fSrc##c0;                                                           \
		const float b##c2 = fSrc##c2 - a##c2 * fSrc##c0;                                                           \
                                                                                                                   \
		const float fDistCorr = sqrt(a##c1*a##c1+a##c2*a##c2+1.0f) * fOutputScale;                                 \
                                                                                                                   \
		float fVal = 0.0f;                                                                                         \
                                                                                                                   \
		float f##c0 = startSlice + 0.5f;                                                                           \
		float f##c1 = a##c1 * (startSlice - 0.5f*dims.iVol##c0 + 0.5f) + b##c1 + 0.5f*dims.iVol##c1 - 0.5f + 0.5f; \
		float f##c2 = a##c2 * (startSlice - 0.5f*dims.iVol##c0 + 0.5f) + b##c2 + 0.5f*dims.iVol##c2 - 0.5f + 0.5f; \
                                                                                                                   \
		for (int s = startSlice; s < endSlice; ++s)                                                                \
		{                                                                                                          \
			fVal += tex3D(gT_coneVolumeTexture, fX, fY, fZ);                                                       \
			f##c0 += 1.0f;                                                                                         \
			f##c1 += a##c1;                                                                                        \
			f##c2 += a##c2;                                                                                        \
		}                                                                                                          \
                                                                                                                   \
		fVal *= fDistCorr;                                                                                         \
		fV += fVal;                                                                                                \
                                                                                                                   \
		}                                                                                                          \
		}                                                                                                          \
                                                                                                                   \
		D_projData[(detectorV*dims.iProjAngles+angle)*projPitch+detectorU] += fV / (dims.iRaysPerDetDim * dims.iRaysPerDetDim);\
	}




 
__global__ void FP_dirX(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_BODY(X,Y,Z)
}

__global__ void FP_dirY(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_BODY(Y,X,Z)
}

__global__ void FP_dirZ(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_BODY(Z,X,Y)
}

 
__global__ void FP_SS_dirX(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_SS_BODY(X,Y,Z)
}

__global__ void FP_SS_dirY(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_SS_BODY(Y,X,Z)
}

__global__ void FP_SS_dirZ(float* D_projData, unsigned int projPitch, unsigned int startSlice, unsigned int startAngle, unsigned int endAngle, const SDimensions3D dims, float fOutputScale)
{
CONE_FP_SS_BODY(Z,X,Y)
}



bool ConeFP_Array(hipArray *D_volArray,
                  hipPitchedPtr D_projData,
                  const SDimensions3D& dims, const SConeProjection* angles,
                  float fOutputScale)
{
	bindVolumeDataTexture(D_volArray);

	// transfer angles to constant memory
	float* tmp = new float[dims.iProjAngles];

#define TRANSFER_TO_CONSTANT(name) do { for (unsigned int i = 0; i < dims.iProjAngles; ++i) tmp[i] = angles[i].f##name ; hipMemcpyToSymbol(HIP_SYMBOL(gC_##name), tmp, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice); } while (0)

	TRANSFER_TO_CONSTANT(SrcX);
	TRANSFER_TO_CONSTANT(SrcY);
	TRANSFER_TO_CONSTANT(SrcZ);
	TRANSFER_TO_CONSTANT(DetSX);
	TRANSFER_TO_CONSTANT(DetSY);
	TRANSFER_TO_CONSTANT(DetSZ);
	TRANSFER_TO_CONSTANT(DetUX);
	TRANSFER_TO_CONSTANT(DetUY);
	TRANSFER_TO_CONSTANT(DetUZ);
	TRANSFER_TO_CONSTANT(DetVX);
	TRANSFER_TO_CONSTANT(DetVY);
	TRANSFER_TO_CONSTANT(DetVZ);

#undef TRANSFER_TO_CONSTANT

	delete[] tmp;

	std::list<hipStream_t> streams;
	dim3 dimBlock(g_detBlockU, g_anglesPerBlock); // region size, angles

	// Run over all angles, grouping them into groups of the same
	// orientation (roughly horizontal vs. roughly vertical).
	// Start a stream of grids for each such group.

	unsigned int blockStart = 0;
	unsigned int blockEnd = 0;
	int blockDirection = 0;

	// timeval t;
	// tic(t);

	for (unsigned int a = 0; a <= dims.iProjAngles; ++a) {
		int dir;
		if (a != dims.iProjAngles) {
			float dX = fabsf(angles[a].fSrcX - (angles[a].fDetSX + dims.iProjU*angles[a].fDetUX*0.5f + dims.iProjV*angles[a].fDetVX*0.5f));
			float dY = fabsf(angles[a].fSrcY - (angles[a].fDetSY + dims.iProjU*angles[a].fDetUY*0.5f + dims.iProjV*angles[a].fDetVY*0.5f));
			float dZ = fabsf(angles[a].fSrcZ - (angles[a].fDetSZ + dims.iProjU*angles[a].fDetUZ*0.5f + dims.iProjV*angles[a].fDetVZ*0.5f));

			if (dX >= dY && dX >= dZ)
				dir = 0;
			else if (dY >= dX && dY >= dZ)
				dir = 1;
			else
				dir = 2;
		}

		if (a == dims.iProjAngles || dir != blockDirection) {
			// block done

			blockEnd = a;
			if (blockStart != blockEnd) {

				dim3 dimGrid(
				             ((dims.iProjU+g_detBlockU-1)/g_detBlockU)*((dims.iProjV+g_detBlockV-1)/g_detBlockV),
(blockEnd-blockStart+g_anglesPerBlock-1)/g_anglesPerBlock);
				// TODO: check if we can't immediately
				//       destroy the stream after use
				hipStream_t stream;
				hipStreamCreate(&stream);
				streams.push_back(stream);

				// printf("angle block: %d to %d, %d (%dx%d, %dx%d)\n", blockStart, blockEnd, blockDirection, dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);

				if (blockDirection == 0) {
					for (unsigned int i = 0; i < dims.iVolX; i += g_blockSlices)
						if (dims.iRaysPerDetDim == 1)
							FP_dirX<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
						else
							FP_SS_dirX<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
				} else if (blockDirection == 1) {
					for (unsigned int i = 0; i < dims.iVolY; i += g_blockSlices)
						if (dims.iRaysPerDetDim == 1)
							FP_dirY<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
						else
							FP_SS_dirY<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
				} else if (blockDirection == 2) {
					for (unsigned int i = 0; i < dims.iVolZ; i += g_blockSlices)
						if (dims.iRaysPerDetDim == 1)
							FP_dirZ<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
						else
							FP_SS_dirZ<<<dimGrid, dimBlock, 0, stream>>>((float*)D_projData.ptr, D_projData.pitch/sizeof(float), i, blockStart, blockEnd, dims, fOutputScale);
				}

			}

			blockDirection = dir;
			blockStart = a;
		}
	}

	for (std::list<hipStream_t>::iterator iter = streams.begin(); iter != streams.end(); ++iter)
		hipStreamDestroy(*iter);

	streams.clear();

	cudaTextForceKernelsCompletion();

	// printf("%f\n", toc(t));

	return true;
}

bool ConeFP(hipPitchedPtr D_volumeData,
            hipPitchedPtr D_projData,
            const SDimensions3D& dims, const SConeProjection* angles,
            float fOutputScale)
{
	// transfer volume to array

	hipArray* cuArray = allocateVolumeArray(dims);
	transferVolumeToArray(D_volumeData, cuArray, dims);

	bool ret = ConeFP_Array(cuArray, D_projData, dims, angles, fOutputScale);

	hipFreeArray(cuArray);

	return ret;
}


}

#ifdef STANDALONE
int main()
{
	SDimensions3D dims;
	dims.iVolX = 256;
	dims.iVolY = 256;
	dims.iVolZ = 256;
	dims.iProjAngles = 32;
	dims.iProjU = 512;
	dims.iProjV = 512;
	dims.iRaysPerDet = 1;

	hipExtent extentV;
	extentV.width = dims.iVolX*sizeof(float);
	extentV.height = dims.iVolY;
	extentV.depth = dims.iVolZ;

	hipPitchedPtr volData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&volData, extentV);

	hipExtent extentP;
	extentP.width = dims.iProjU*sizeof(float);
	extentP.height = dims.iProjV;
	extentP.depth = dims.iProjAngles;

	hipPitchedPtr projData; // pitch, ptr, xsize, ysize

	hipMalloc3D(&projData, extentP);
	hipMemset3D(projData, 0, extentP);

	float* slice = new float[256*256];
	hipPitchedPtr ptr;
	ptr.ptr = slice;
	ptr.pitch = 256*sizeof(float);
	ptr.xsize = 256*sizeof(float);
	ptr.ysize = 256;

	for (unsigned int i = 0; i < 256*256; ++i)
		slice[i] = 1.0f;
	for (unsigned int i = 0; i < 256; ++i) {
		hipExtent extentS;
		extentS.width = dims.iVolX*sizeof(float);
		extentS.height = dims.iVolY;
		extentS.depth = 1;
		hipPos sp = { 0, 0, 0 };
		hipPos dp = { 0, 0, i };
		hipMemcpy3DParms p;
		p.srcArray = 0;
		p.srcPos = sp;
		p.srcPtr = ptr;
		p.dstArray = 0;
		p.dstPos = dp;
		p.dstPtr = volData;
		p.extent = extentS;
		p.kind = hipMemcpyHostToDevice;
		hipError_t err = hipMemcpy3D(&p);
		assert(!err);
	}


	SConeProjection angle[32];
	angle[0].fSrcX = -1536;
	angle[0].fSrcY = 0;
	angle[0].fSrcZ = 200;

	angle[0].fDetSX = 512;
	angle[0].fDetSY = -256;
	angle[0].fDetSZ = -256;

	angle[0].fDetUX = 0;
	angle[0].fDetUY = 1;
	angle[0].fDetUZ = 0;

	angle[0].fDetVX = 0;
	angle[0].fDetVY = 0;
	angle[0].fDetVZ = 1;

#define ROTATE0(name,i,alpha) do { angle[i].f##name##X = angle[0].f##name##X * cos(alpha) - angle[0].f##name##Y * sin(alpha); angle[i].f##name##Y = angle[0].f##name##X * sin(alpha) + angle[0].f##name##Y * cos(alpha); } while(0)
	for (int i = 1; i < 32; ++i) {
		angle[i] = angle[0];
		ROTATE0(Src, i, i*1*M_PI/180);
		ROTATE0(DetS, i, i*1*M_PI/180);
		ROTATE0(DetU, i, i*1*M_PI/180);
		ROTATE0(DetV, i, i*1*M_PI/180);
	}
#undef ROTATE0

	astraCUDA3d::ConeFP(volData, projData, dims, angle, 1.0f);

	float* buf = new float[512*512];

	hipMemcpy(buf, ((float*)projData.ptr)+512*512*8, 512*512*sizeof(float), hipMemcpyDeviceToHost);

	printf("%d %d %d\n", projData.pitch, projData.xsize, projData.ysize);

	saveImage("proj.png", 512, 512, buf);
	

}
#endif
