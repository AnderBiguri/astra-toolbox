#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "util.h"
#include "par_fp.h"
#include "fan_fp.h"
#include "par_bp.h"
#include "fan_bp.h"
#include "arith.h"
#include "astra.h"

#include "fft.h"

#include <fstream>
#include <hip/hip_runtime.h>

#include "../../include/astra/Logger.h"

// For fan beam FBP weighting
#include "../3d/fdk.h"

using namespace astraCUDA;
using namespace std;


namespace astra {

enum CUDAProjectionType {
	PROJ_PARALLEL,
	PROJ_FAN
};


class AstraFBP_internal {
public:
	SDimensions dims;
	float* angles;
	float* TOffsets;

	float fOriginSourceDistance;
	float fOriginDetectorDistance;

	float fPixelSize;

	bool bFanBeam;
	bool bShortScan;

	bool initialized;
	bool setStartReconstruction;

	float* D_sinoData;
	unsigned int sinoPitch;

	float* D_volumeData;
	unsigned int volumePitch;

	hipfftComplex * m_pDevFilter;
};

AstraFBP::AstraFBP()
{
	pData = new AstraFBP_internal();

	pData->angles = 0;
	pData->D_sinoData = 0;
	pData->D_volumeData = 0;

	pData->dims.iVolWidth = 0;
	pData->dims.iProjAngles = 0;
	pData->dims.fDetScale = 1.0f;
	pData->dims.iRaysPerDet = 1;
	pData->dims.iRaysPerPixelDim = 1;

	pData->initialized = false;
	pData->setStartReconstruction = false;

	pData->m_pDevFilter = NULL;
}

AstraFBP::~AstraFBP()
{
	delete[] pData->angles;
	pData->angles = 0;

	delete[] pData->TOffsets;
	pData->TOffsets = 0;

	hipFree(pData->D_sinoData);
	pData->D_sinoData = 0;

	hipFree(pData->D_volumeData);
	pData->D_volumeData = 0;

	if(pData->m_pDevFilter != NULL)
	{
		freeComplexOnDevice(pData->m_pDevFilter);
		pData->m_pDevFilter = NULL;
	}

	delete pData;
	pData = 0;
}

bool AstraFBP::setReconstructionGeometry(unsigned int iVolWidth,
                                          unsigned int iVolHeight,
                                          float fPixelSize)
{
	if (pData->initialized)
		return false;

	pData->dims.iVolWidth = iVolWidth;
	pData->dims.iVolHeight = iVolHeight;

	pData->fPixelSize = fPixelSize;

	return (iVolWidth > 0 && iVolHeight > 0 && fPixelSize > 0.0f);
}

bool AstraFBP::setProjectionGeometry(unsigned int iProjAngles,
                                      unsigned int iProjDets,
                                      const float* pfAngles,
                                      float fDetSize)
{
	if (pData->initialized)
		return false;

	pData->dims.iProjAngles = iProjAngles;
	pData->dims.iProjDets = iProjDets;
	pData->dims.fDetScale = fDetSize / pData->fPixelSize;

	if (iProjAngles == 0 || iProjDets == 0 || pfAngles == 0)
		return false;

	pData->angles = new float[iProjAngles];
	memcpy(pData->angles, pfAngles, iProjAngles * sizeof(pfAngles[0]));

	pData->bFanBeam = false;

	return true;
}

bool AstraFBP::setFanGeometry(unsigned int iProjAngles,
                              unsigned int iProjDets,
                              const float* pfAngles,
                              float fOriginSourceDistance,
                              float fOriginDetectorDistance,
                              float fDetSize,
                              bool bShortScan)
{
	// Slightly abusing setProjectionGeometry for this...
	if (!setProjectionGeometry(iProjAngles, iProjDets, pfAngles, fDetSize))
		return false;

	pData->fOriginSourceDistance = fOriginSourceDistance;
	pData->fOriginDetectorDistance = fOriginDetectorDistance;

	pData->bFanBeam = true;
	pData->bShortScan = bShortScan;

	return true;
}


bool AstraFBP::setPixelSuperSampling(unsigned int iPixelSuperSampling)
{
	if (pData->initialized)
		return false;

	if (iPixelSuperSampling == 0)
		return false;

	pData->dims.iRaysPerPixelDim = iPixelSuperSampling;

	return true;
}


bool AstraFBP::setTOffsets(const float* pfTOffsets)
{
	if (pData->initialized)
		return false;

	if (pfTOffsets == 0)
		return false;

	pData->TOffsets = new float[pData->dims.iProjAngles];
	memcpy(pData->TOffsets, pfTOffsets, pData->dims.iProjAngles * sizeof(pfTOffsets[0]));

	return true;
}

bool AstraFBP::init(int iGPUIndex)
{
	if (pData->initialized)
	{
		return false;
	}

	if (pData->dims.iProjAngles == 0 || pData->dims.iVolWidth == 0)
	{
		return false;
	}

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
		{
			return false;
		}
	}

	bool ok = allocateVolumeData(pData->D_volumeData, pData->volumePitch, pData->dims);
	if (!ok)
	{
		return false;
	}

	ok = allocateProjectionData(pData->D_sinoData, pData->sinoPitch, pData->dims);
	if (!ok)
	{
		hipFree(pData->D_volumeData);
		pData->D_volumeData = 0;
		return false;
	}

	pData->initialized = true;

	return true;
}

bool AstraFBP::setSinogram(const float* pfSinogram,
                            unsigned int iSinogramPitch)
{
	if (!pData->initialized)
		return false;
	if (!pfSinogram)
		return false;

	bool ok = copySinogramToDevice(pfSinogram, iSinogramPitch,
	                               pData->dims.iProjDets,
	                               pData->dims.iProjAngles,
	                               pData->D_sinoData, pData->sinoPitch);
	if (!ok)
		return false;

	// rescale sinogram to adjust for pixel size
	processVol<opMul>(pData->D_sinoData,
	                       1.0f/(pData->fPixelSize*pData->fPixelSize),
	                       pData->sinoPitch,
	                       pData->dims.iProjDets, pData->dims.iProjAngles);

	pData->setStartReconstruction = false;

	return true;
}

static int calcNextPowerOfTwo(int _iValue)
{
	int iOutput = 1;

	while(iOutput < _iValue)
	{
		iOutput *= 2;
	}

	return iOutput;
}

bool AstraFBP::run()
{
	if (!pData->initialized)
	{
		return false;
	}

	zeroVolumeData(pData->D_volumeData, pData->volumePitch, pData->dims);

	bool ok = false;

	if (pData->bFanBeam) {
		// Call FDK_PreWeight to handle fan beam geometry. We treat
		// this as a cone beam setup of a single slice:

		// TODO: TOffsets...

		// We create a fake hipPitchedPtr
		hipPitchedPtr tmp;
		tmp.ptr = pData->D_sinoData;
		tmp.pitch = pData->sinoPitch * sizeof(float);
		tmp.xsize = pData->dims.iProjDets;
		tmp.ysize = pData->dims.iProjAngles;
		// and a fake Dimensions3D
		astraCUDA3d::SDimensions3D dims3d;
		dims3d.iVolX = pData->dims.iVolWidth;
		dims3d.iVolY = pData->dims.iVolHeight;
		dims3d.iVolZ = 1;
		dims3d.iProjAngles = pData->dims.iProjAngles;
		dims3d.iProjU = pData->dims.iProjDets;
		dims3d.iProjV = 1;
		dims3d.iRaysPerDetDim = dims3d.iRaysPerVoxelDim = 1;

		astraCUDA3d::FDK_PreWeight(tmp, pData->fOriginSourceDistance,
		              pData->fOriginDetectorDistance, 0.0f, 0.0f,
		              pData->dims.fDetScale, 1.0f, // TODO: Are these correct?
		              pData->bShortScan, dims3d, pData->angles);
	}

	if (pData->m_pDevFilter) {

		int iFFTRealDetCount = calcNextPowerOfTwo(2 * pData->dims.iProjDets);
		int iFFTFourDetCount = calcFFTFourSize(iFFTRealDetCount);

		hipfftComplex * pDevComplexSinogram = NULL;

		allocateComplexOnDevice(pData->dims.iProjAngles, iFFTFourDetCount, &pDevComplexSinogram);

		runCudaFFT(pData->dims.iProjAngles, pData->D_sinoData, pData->sinoPitch, pData->dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount, pDevComplexSinogram);

		applyFilter(pData->dims.iProjAngles, iFFTFourDetCount, pDevComplexSinogram, pData->m_pDevFilter);

		runCudaIFFT(pData->dims.iProjAngles, pDevComplexSinogram, pData->D_sinoData, pData->sinoPitch, pData->dims.iProjDets, iFFTRealDetCount, iFFTFourDetCount);

		freeComplexOnDevice(pDevComplexSinogram);

	}

	if (pData->bFanBeam) {
		// TODO: TOffsets?
		// TODO: Remove this code duplication with CudaReconstructionAlgorithm
		SFanProjection* projs;
		projs = new SFanProjection[pData->dims.iProjAngles];

		float fSrcX0 = 0.0f;
		float fSrcY0 = -pData->fOriginSourceDistance / pData->fPixelSize;
		float fDetUX0 = pData->dims.fDetScale;
		float fDetUY0 = 0.0f;
		float fDetSX0 = pData->dims.iProjDets * fDetUX0 / -2.0f;
		float fDetSY0 = pData->fOriginDetectorDistance / pData->fPixelSize;

#define ROTATE0(name,i,alpha) do { projs[i].f##name##X = f##name##X0 * cos(alpha) - f##name##Y0 * sin(alpha); projs[i].f##name##Y = f##name##X0 * sin(alpha) + f##name##Y0 * cos(alpha); } while(0)
		for (unsigned int i = 0; i < pData->dims.iProjAngles; ++i) {
			ROTATE0(Src, i, pData->angles[i]);
			ROTATE0(DetS, i, pData->angles[i]);
			ROTATE0(DetU, i, pData->angles[i]);
		}

#undef ROTATE0
		ok = FanBP_FBPWeighted(pData->D_volumeData, pData->volumePitch, pData->D_sinoData, pData->sinoPitch, pData->dims, projs);

		delete[] projs;

	} else {
		ok = BP(pData->D_volumeData, pData->volumePitch, pData->D_sinoData, pData->sinoPitch, pData->dims, pData->angles, pData->TOffsets);
	}
	if(!ok)
	{
		return false;
	}

	processVol<opMul>(pData->D_volumeData,
	                      (M_PI / 2.0f) / (float)pData->dims.iProjAngles,
	                      pData->volumePitch,
	                      pData->dims.iVolWidth, pData->dims.iVolHeight);

	return true;
}

bool AstraFBP::getReconstruction(float* pfReconstruction, unsigned int iReconstructionPitch) const
{
	if (!pData->initialized)
		return false;

	bool ok = copyVolumeFromDevice(pfReconstruction, iReconstructionPitch,
	                               pData->dims.iVolWidth,
	                               pData->dims.iVolHeight,
	                               pData->D_volumeData, pData->volumePitch);
	if (!ok)
		return false;

	return true;
}

int AstraFBP::calcFourierFilterSize(int _iDetectorCount)
{
	int iFFTRealDetCount = calcNextPowerOfTwo(2 * _iDetectorCount);
	int iFreqBinCount = calcFFTFourSize(iFFTRealDetCount);

	// CHECKME: Matlab makes this at least 64. Do we also need to?
	return iFreqBinCount;
}

bool AstraFBP::setFilter(E_FBPFILTER _eFilter, const float * _pfHostFilter /* = NULL */, int _iFilterWidth /* = 0 */, float _fD /* = 1.0f */, float _fFilterParameter /* = -1.0f */)
{
	if(pData->m_pDevFilter != 0)
	{
		freeComplexOnDevice(pData->m_pDevFilter);
		pData->m_pDevFilter = 0;
	}

	if (_eFilter == FILTER_NONE)
		return true; // leave pData->m_pDevFilter set to 0


	int iFFTRealDetCount = calcNextPowerOfTwo(2 * pData->dims.iProjDets);
	int iFreqBinCount = calcFFTFourSize(iFFTRealDetCount);

	hipfftComplex * pHostFilter = new hipfftComplex[pData->dims.iProjAngles * iFreqBinCount];
	memset(pHostFilter, 0, sizeof(hipfftComplex) * pData->dims.iProjAngles * iFreqBinCount);

	allocateComplexOnDevice(pData->dims.iProjAngles, iFreqBinCount, &(pData->m_pDevFilter));

	switch(_eFilter)
	{
		case FILTER_NONE:
			// handled above
			break;
		case FILTER_RAMLAK:
		case FILTER_SHEPPLOGAN:
		case FILTER_COSINE:
		case FILTER_HAMMING:
		case FILTER_HANN:
		case FILTER_TUKEY:
		case FILTER_LANCZOS:
		case FILTER_TRIANGULAR:
		case FILTER_GAUSSIAN:
		case FILTER_BARTLETTHANN:
		case FILTER_BLACKMAN:
		case FILTER_NUTTALL:
		case FILTER_BLACKMANHARRIS:
		case FILTER_BLACKMANNUTTALL:
		case FILTER_FLATTOP:
		case FILTER_PARZEN:
		{
			genFilter(_eFilter, _fD, pData->dims.iProjAngles, pHostFilter, iFFTRealDetCount, iFreqBinCount, _fFilterParameter);
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);

			break;
		}
		case FILTER_PROJECTION:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				float fValue = _pfHostFilter[iFreqBinIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);
			break;
		}
		case FILTER_SINOGRAM:
		{
			// make sure the offered filter has the correct size
			assert(_iFilterWidth == iFreqBinCount);

			for(int iFreqBinIndex = 0; iFreqBinIndex < iFreqBinCount; iFreqBinIndex++)
			{
				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iFreqBinIndex + iProjectionIndex * _iFilterWidth];

					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].x = fValue;
					pHostFilter[iFreqBinIndex + iProjectionIndex * iFreqBinCount].y = 0.0f;
				}
			}
			uploadComplexArrayToDevice(pData->dims.iProjAngles, iFreqBinCount, pHostFilter, pData->m_pDevFilter);
			break;
		}
		case FILTER_RPROJECTION:
		{
			int iProjectionCount = pData->dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float * pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;

			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;
				float fValue = _pfHostFilter[iDetectorIndex];

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, pData->m_pDevFilter);

			hipFree(pfDevRealFilter);

			break;
		}
		case FILTER_RSINOGRAM:
		{
			int iProjectionCount = pData->dims.iProjAngles;
			int iRealFilterElementCount = iProjectionCount * iFFTRealDetCount;
			float* pfHostRealFilter = new float[iRealFilterElementCount];
			memset(pfHostRealFilter, 0, sizeof(float) * iRealFilterElementCount);

			int iUsedFilterWidth = min(_iFilterWidth, iFFTRealDetCount);
			int iStartFilterIndex = (_iFilterWidth - iUsedFilterWidth) / 2;
			int iMaxFilterIndex = iStartFilterIndex + iUsedFilterWidth;

			int iFilterShiftSize = _iFilterWidth / 2;
			
			for(int iDetectorIndex = iStartFilterIndex; iDetectorIndex < iMaxFilterIndex; iDetectorIndex++)
			{
				int iFFTInFilterIndex = (iDetectorIndex + iFFTRealDetCount - iFilterShiftSize) % iFFTRealDetCount;

				for(int iProjectionIndex = 0; iProjectionIndex < (int)pData->dims.iProjAngles; iProjectionIndex++)
				{
					float fValue = _pfHostFilter[iDetectorIndex + iProjectionIndex * _iFilterWidth];
					pfHostRealFilter[iFFTInFilterIndex + iProjectionIndex * iFFTRealDetCount] = fValue;
				}
			}

			float* pfDevRealFilter = NULL;
			hipMalloc((void **)&pfDevRealFilter, sizeof(float) * iRealFilterElementCount); // TODO: check for errors
			hipMemcpy(pfDevRealFilter, pfHostRealFilter, sizeof(float) * iRealFilterElementCount, hipMemcpyHostToDevice);
			delete[] pfHostRealFilter;

			runCudaFFT(iProjectionCount, pfDevRealFilter, iFFTRealDetCount, iFFTRealDetCount, iFFTRealDetCount, iFreqBinCount, pData->m_pDevFilter);

			hipFree(pfDevRealFilter);

			break;
		}
		default:
		{
			fprintf(stderr, "AstraFBP::setFilter: Unknown filter type requested");
			delete [] pHostFilter;
			return false;
		}
	}

	delete [] pHostFilter;

	return true;
}

BPalgo::BPalgo()
{

}

BPalgo::~BPalgo()
{

}

bool BPalgo::init()
{
	return true;
}

bool BPalgo::iterate(unsigned int)
{
	// TODO: This zeroVolume makes an earlier memcpy of D_volumeData redundant
	zeroVolumeData(D_volumeData, volumePitch, dims);
	callBP(D_volumeData, volumePitch, D_sinoData, sinoPitch);
	return true;
}

float BPalgo::computeDiffNorm()
{
	float *D_projData;
	unsigned int projPitch;

	allocateProjectionData(D_projData, projPitch, dims);

	hipMemcpy2D(D_projData, sizeof(float)*projPitch, D_sinoData, sizeof(float)*sinoPitch, sizeof(float)*dims.iProjDets, dims.iProjAngles, hipMemcpyDeviceToDevice);
	callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles);

	hipFree(D_projData);

	return sqrt(s);
}


bool astraCudaFP(const float* pfVolume, float* pfSinogram,
                 unsigned int iVolWidth, unsigned int iVolHeight,
                 unsigned int iProjAngles, unsigned int iProjDets,
                 const float *pfAngles, const float *pfOffsets,
                 float fDetSize, unsigned int iDetSuperSampling,
                 int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pfAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;
	dims.fDetScale = fDetSize;

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims.iVolWidth, dims.iVolHeight,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);
	ok = FP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pfAngles, pfOffsets, 1.0f);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims.iProjDets,
	                            dims.iProjAngles,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);
	return true;
}

bool astraCudaFanFP(const float* pfVolume, float* pfSinogram,
                    unsigned int iVolWidth, unsigned int iVolHeight,
                    unsigned int iProjAngles, unsigned int iProjDets,
                    const float *pfAngles, float fOriginSourceDistance,
                    float fOriginDetectorDistance, float fPixelSize,
                    float fDetSize,
                    unsigned int iDetSuperSampling,
                    int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pfAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims.iVolWidth, dims.iVolHeight,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);

	// TODO: Turn this geometry conversion into a util function
	SFanProjection* projs = new SFanProjection[dims.iProjAngles];

	float fSrcX0 = 0.0f;
	float fSrcY0 = -fOriginSourceDistance / fPixelSize;
	float fDetUX0 = fDetSize / fPixelSize;
	float fDetUY0 = 0.0f;
	float fDetSX0 = dims.iProjDets * fDetUX0 / -2.0f;
	float fDetSY0 = fOriginDetectorDistance / fPixelSize;

#define ROTATE0(name,i,alpha) do { projs[i].f##name##X = f##name##X0 * cos(alpha) - f##name##Y0 * sin(alpha); projs[i].f##name##Y = f##name##X0 * sin(alpha) + f##name##Y0 * cos(alpha); } while(0)
	for (int i = 0; i < dims.iProjAngles; ++i) {
		ROTATE0(Src, i, pfAngles[i]);
		ROTATE0(DetS, i, pfAngles[i]);
		ROTATE0(DetU, i, pfAngles[i]);
	}

#undef ROTATE0

	ok = FanFP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, projs, 1.0f);
	delete[] projs;

	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims.iProjDets,
	                            dims.iProjAngles,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);

	return true;

}


bool astraCudaFanFP(const float* pfVolume, float* pfSinogram,
                    unsigned int iVolWidth, unsigned int iVolHeight,
                    unsigned int iProjAngles, unsigned int iProjDets,
                    const SFanProjection *pAngles,
                    unsigned int iDetSuperSampling,
                    int iGPUIndex)
{
	SDimensions dims;

	if (iProjAngles == 0 || iProjDets == 0 || pAngles == 0)
		return false;

	dims.iProjAngles = iProjAngles;
	dims.iProjDets = iProjDets;
	dims.fDetScale = 1.0f; // TODO?

	if (iDetSuperSampling == 0)
		return false;

	dims.iRaysPerDet = iDetSuperSampling;

	if (iVolWidth <= 0 || iVolHeight <= 0)
		return false;

	dims.iVolWidth = iVolWidth;
	dims.iVolHeight = iVolHeight;

	if (iGPUIndex != -1) {
		hipSetDevice(iGPUIndex);
		hipError_t err = hipGetLastError();

		// Ignore errors caused by calling hipSetDevice multiple times
		if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
			return false;
	}

	bool ok;

	float* D_volumeData;
	unsigned int volumePitch;

	ok = allocateVolumeData(D_volumeData, volumePitch, dims);
	if (!ok)
		return false;

	float* D_sinoData;
	unsigned int sinoPitch;

	ok = allocateProjectionData(D_sinoData, sinoPitch, dims);
	if (!ok) {
		hipFree(D_volumeData);
		return false;
	}

	ok = copyVolumeToDevice(pfVolume, dims.iVolWidth,
	                        dims.iVolWidth, dims.iVolHeight,
	                        D_volumeData, volumePitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	zeroProjectionData(D_sinoData, sinoPitch, dims);

	ok = FanFP(D_volumeData, volumePitch, D_sinoData, sinoPitch, dims, pAngles, 1.0f);

	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	ok = copySinogramFromDevice(pfSinogram, dims.iProjDets,
	                            dims.iProjDets,
	                            dims.iProjAngles,
	                            D_sinoData, sinoPitch);
	if (!ok) {
		hipFree(D_volumeData);
		hipFree(D_sinoData);
		return false;
	}

	hipFree(D_volumeData);
	hipFree(D_sinoData);

	return true;

}


}
