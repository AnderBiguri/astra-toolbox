#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include "util.h"

namespace astraCUDA {

bool copyVolumeToDevice(const float* in_data, unsigned int in_pitch,
		unsigned int width, unsigned int height,
		float* outD_data, unsigned int out_pitch)
{
	// TODO: a full memset isn't necessary. Only the edges.
	hipError_t err;
	err = hipMemset2D(outD_data, sizeof(float)*out_pitch, 0, sizeof(float)*(width+2), height+2);
	ASTRA_CUDA_ASSERT(err);
	err = hipMemcpy2D(outD_data + out_pitch + 1, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice);
	ASTRA_CUDA_ASSERT(err);
	assert(err == hipSuccess);
	return true;
}

bool copyVolumeFromDevice(float* out_data, unsigned int out_pitch,
		unsigned int width, unsigned int height,
		float* inD_data, unsigned int in_pitch)
{
	hipError_t err = hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data + (in_pitch + 1), sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost);
	ASTRA_CUDA_ASSERT(err);
	return true;
}


bool copySinogramFromDevice(float* out_data, unsigned int out_pitch,
		unsigned int width, unsigned int height,
		float* inD_data, unsigned int in_pitch)
{   
	hipError_t err = hipMemcpy2D(out_data, sizeof(float)*out_pitch, inD_data + 1, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyDeviceToHost);
	ASTRA_CUDA_ASSERT(err);
	return true;
}

bool copySinogramToDevice(const float* in_data, unsigned int in_pitch,
		unsigned int width, unsigned int height,
		float* outD_data, unsigned int out_pitch)
{   
	// TODO: a full memset isn't necessary. Only the edges.
	hipError_t err;
	err = hipMemset2D(outD_data, sizeof(float)*out_pitch, 0, (width+2)*sizeof(float), height);
	ASTRA_CUDA_ASSERT(err);
	err = hipMemcpy2D(outD_data + 1, sizeof(float)*out_pitch, in_data, sizeof(float)*in_pitch, sizeof(float)*width, height, hipMemcpyHostToDevice);
	ASTRA_CUDA_ASSERT(err);
	return true;
}


bool allocateVolume(float*& ptr, unsigned int width, unsigned int height, unsigned int& pitch)
{
	size_t p;
	hipError_t ret = hipMallocPitch((void**)&ptr, &p, sizeof(float)*width, height);
	if (ret != hipSuccess) {
		reportCudaError(ret);
		fprintf(stderr, "Failed to allocate %dx%d GPU buffer\n", width, height);
		return false;
	}

	assert(p % sizeof(float) == 0);

	pitch = p / sizeof(float);

	return true;
}

void zeroVolume(float* data, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipError_t err;
	err = hipMemset2D(data, sizeof(float)*pitch, 0, sizeof(float)*width, height);
	ASTRA_CUDA_ASSERT(err);
}


template <unsigned int blockSize>
__global__ void reduce1D(float *g_idata, float *g_odata, unsigned int n)
{
	extern __shared__ float sdata[];
	unsigned int tid = threadIdx.x;

	unsigned int i = blockIdx.x*(blockSize*2) + tid;
	unsigned int gridSize = blockSize*gridDim.x;
	sdata[tid] = 0;
	while (i < n) { sdata[tid] += g_idata[i]; i += gridSize; }
	__syncthreads();
	if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
	if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
	if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }
	if (tid < 32) {
		volatile float* smem = sdata;
		if (blockSize >= 64) smem[tid] += smem[tid + 32];
		if (blockSize >= 32) smem[tid] += smem[tid + 16];
		if (blockSize >= 16) smem[tid] += smem[tid + 8];
		if (blockSize >= 8) smem[tid] += smem[tid + 4];
		if (blockSize >= 4) smem[tid] += smem[tid + 2];
		if (blockSize >= 2) smem[tid] += smem[tid + 1];
	}
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void reduce2D(float *g_idata, float *g_odata,
                         unsigned int pitch,
                         unsigned int nx, unsigned int ny,
                         unsigned int padX, unsigned int padY)
{
	extern __shared__ float sdata[];
	const unsigned int tidx = threadIdx.x;
	const unsigned int tidy = threadIdx.y;
	const unsigned int tid = tidy * 16 + tidx;

	unsigned int x = blockIdx.x*16 + tidx;
	unsigned int y = blockIdx.y*16 + tidy;

	sdata[tid] = 0;

	if (x >= padX && x < padX + nx) {

		while (y < padY + ny) {
			if (y >= padY)
				sdata[tid] += (g_idata[pitch*y+x] * g_idata[pitch*y+x]);
			y += 16 * gridDim.y;
		}

	}

	__syncthreads();

	if (tid < 128)
		sdata[tid] += sdata[tid + 128];
	__syncthreads();

	if (tid < 64)
		sdata[tid] += sdata[tid + 64];
	__syncthreads();

	if (tid < 32) { // 32 is warp size
		volatile float* smem = sdata;
		smem[tid] += smem[tid + 32];
		smem[tid] += smem[tid + 16];
		smem[tid] += smem[tid + 8];
		smem[tid] += smem[tid + 4];
		smem[tid] += smem[tid + 2];
		smem[tid] += smem[tid + 1];
	} 

	if (tid == 0)
		g_odata[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
}

float dotProduct2D(float* D_data, unsigned int pitch,
                   unsigned int width, unsigned int height,
                   unsigned int padX, unsigned int padY)
{
	unsigned int bx = ((width+padX) + 15) / 16;
	unsigned int by = ((height+padY) + 127) / 128;
	unsigned int shared_mem2 = sizeof(float) * 16 * 16;

	dim3 dimBlock2(16, 16);
	dim3 dimGrid2(bx, by);

	float* D_buf;
	hipMalloc(&D_buf, sizeof(float) * (bx * by + 1) );

	// Step 1: reduce 2D from image to a single vector, taking sum of squares

	reduce2D<<< dimGrid2, dimBlock2, shared_mem2>>>(D_data, D_buf, pitch, width, height, padX, padY);
	cudaTextForceKernelsCompletion();

	// Step 2: reduce 1D: add up elements in vector
	if (bx * by > 512)
		reduce1D<512><<< 1, 512, sizeof(float)*512>>>(D_buf, D_buf+(bx*by), bx*by);
	else if (bx * by > 128)
		reduce1D<128><<< 1, 128, sizeof(float)*128>>>(D_buf, D_buf+(bx*by), bx*by);
	else if (bx * by > 32)
		reduce1D<32><<< 1, 32, sizeof(float)*32*2>>>(D_buf, D_buf+(bx*by), bx*by);
	else if (bx * by > 8)
		reduce1D<8><<< 1, 8, sizeof(float)*8*2>>>(D_buf, D_buf+(bx*by), bx*by);
	else
		reduce1D<1><<< 1, 1, sizeof(float)*1*2>>>(D_buf, D_buf+(bx*by), bx*by);

	float x;
	hipMemcpy(&x, D_buf+(bx*by), 4, hipMemcpyDeviceToHost);

	cudaTextForceKernelsCompletion();

	hipFree(D_buf);

	return x;
}


bool cudaTextForceKernelsCompletion()
{
	hipError_t returnedCudaError = hipDeviceSynchronize();

	if(returnedCudaError != hipSuccess) {
		fprintf(stderr, "Failed to force completion of cuda kernels: %d: %s.\n", returnedCudaError, hipGetErrorString(returnedCudaError));
		return false;
	}

	return true;
}

void reportCudaError(hipError_t err)
{
	if(err != hipSuccess)
		fprintf(stderr, "CUDA error %d: %s.\n", err, hipGetErrorString(err));
}



}
