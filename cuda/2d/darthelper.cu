#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include "util.h"
#include "darthelper.h"
#include <cassert>

namespace astraCUDA {

// CUDA function for the selection of ROI
__global__ void devRoiSelect(float* in, float radius, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	float x = (float)(threadIdx.x + 16*blockIdx.x);
	float y = (float)(threadIdx.y + 16*blockIdx.y);

	float w = (width-1.0f)*0.5f;
	float h = (height-1.0f)*0.5f;

	if ((x-w)*(x-w) + (y-h)*(y-h) > radius * radius * 0.25f) 
	{
		float* d = (float*)in;
		unsigned int o = (y+padY)*pitch+x+padX; 
		d[o] = 0.0f;
	}
}

void roiSelect(float* out, float radius, unsigned int width, unsigned int height)
{
	float* D_data;

	unsigned int pitch;
	allocateVolume(D_data, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_data, pitch);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);
	devRoiSelect<<<gridSize, blockSize>>>(D_data, radius, pitch, width, height, 1, 1);

	copyVolumeFromDevice(out, width, width, height, D_data, pitch);

	hipFree(D_data);
}




// CUDA function for the masking of DART with a radius == 1
__global__ void devDartMask(float* mask, const float* in, unsigned int conn, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)mask;

		unsigned int o2 = (y+padY)*pitch+x+padX; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		if ((conn == 8 && // 8-connected
		        (d[o1 - 1] != d[o2] || d[o1] != d[o2] || d[o1 + 1] != d[o2] || 
		         d[o2 - 1] != d[o2] ||                   d[o2 + 1] != d[o2] ||
				 d[o3 - 1] != d[o2] || d[o3] != d[o2] || d[o3 + 1] != d[o2] ))
			|| 
			(conn == 4 && // 4-connected
		        (                      d[o1] != d[o2] ||                      
		         d[o2 - 1] != d[o2] ||                  d[o3 + 1] != d[o2] ||
				                       d[o3] != d[o2]                      )))
		{
			m[o2] = 1.0f;
		}
	}
}


// CUDA function for the masking of DART with a radius > 1
__global__ void devDartMaskRadius(float* mask, const float* in, unsigned int conn, unsigned int radius, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius) 
	{
		float* d = (float*)in;
		float* m = (float*)mask;

		int r = radius;

		// o2: index of the current center pixel
		int o2 = (y+padY)*pitch+x+padX;

		if (conn == 8) // 8-connected
		{
			for (int row = -r; row <= r; row++) 
			{
				int o1 = (y+padY+row)*pitch+x+padX; 
				for (int col = -r; col <= r; col++) 
				{
					if (d[o1 + col] != d[o2]) {m[o2] = 1.0f; return;}
				}
			}
		}
		else if (conn == 4) // 4-connected
		{
			// horizontal
			unsigned int o1 = (y+padY)*pitch+x+padX; 
			for (int col = -r; col <= r; col++) 
			{
				if (d[o1 + col] != d[o2]) {m[o2] = 1.0f; return;}
			}

			// vertical
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+padY+row)*pitch+x+padX; 
				if (d[o1] != d[o2]) {m[o2] = 1.0f; return;}
			}
		}
	}
}


// CUDA function for the masking of ADART with a radius == 1
__global__ void devADartMask(float* mask, const float* in, unsigned int conn, unsigned int threshold, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)mask;

		unsigned int o2 = (y+padY)*pitch+x+padX; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		if (conn == 8)
		{
			if (d[o1 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o1    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o1 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o2 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o2 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o3 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		}
		else if (conn == 4)
		{
			if (d[o1    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		    if (d[o2 - 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o2 + 1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			if (d[o3    ] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
		}
	}
}


// CUDA function for the masking of ADART with a radius > 1
__global__ void devADartMaskRadius(float* mask, const float* in, unsigned int conn, unsigned int radius, unsigned int threshold, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius)
	{
		float* d = (float*)in;
		float* m = (float*)mask;
	
		int r = radius;

		unsigned int o2 = (y+padY)*pitch+x+padX; // On this row.

		if (conn == 8)
		{
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+padY+row)*pitch+x+padX; 
				for (int col = -r; col <= r; col++) 
				{
					if (d[o1+col] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
				}
			}
		}
		else if (conn == 4)
		{
			// horizontal
			for (int col = -r; col <= r; col++) 
			{
				if (d[o2+col] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			}

			// vertical
			for (int row = -r; row <= r; row++) 
			{
				unsigned int o1 = (y+padY+row)*pitch+x+padX; 
				if (d[o1] != d[o2] && --threshold == 0) {m[o2] = 1.0f; return;}
			}
		}
	}
}


void dartMask(float* mask, const float* segmentation, unsigned int conn, unsigned int radius, unsigned int threshold, unsigned int width, unsigned int height)
{
	float* D_segmentationData;
	float* D_maskData;

	unsigned int pitch;
	allocateVolume(D_segmentationData, width+2, height+2, pitch);
	copyVolumeToDevice(segmentation, width, width, height, D_segmentationData, pitch);

	allocateVolume(D_maskData, width+2, height+2, pitch);
	zeroVolume(D_maskData, pitch, width+2, height+2);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	if (threshold == 1 && radius == 1)
		devDartMask<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, pitch, width, height, 1, 1);
	else if (threshold > 1 && radius == 1)
		devADartMask<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, threshold, pitch, width, height, 1, 1);
	else if (threshold == 1 && radius > 1)
		devDartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, pitch, width, height, 1, 1);
	else 
		devADartMaskRadius<<<gridSize, blockSize>>>(D_maskData, D_segmentationData, conn, radius, threshold, pitch, width, height, 1, 1);

	copyVolumeFromDevice(mask, width, width, height, D_maskData, pitch);

	hipFree(D_segmentationData);
	hipFree(D_maskData);

}


__global__ void devDartSmoothingRadius(float* out, const float* in, float b, unsigned int radius, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > radius-1 && x < width - radius && y > radius-1 && y < height - radius)
	{
		float* d = (float*)in;
		float* m = (float*)out;

		unsigned int o2 = (y+padY)*pitch+x+padX;
		int r = radius;
		float res = -d[o2];

		for (int row = -r; row < r; row++) 
		{
			unsigned int o1 = (y+padY+row)*pitch+x+padX; 
			for (int col = -r; col <= r; col++) 
			{
				res += d[o1+col];
			}
		}

		res *= b / 4*r*(r+1);
		res += (1.0f-b) * d[o2];

		m[o2] = res;
	}
}


__global__ void devDartSmoothing(float* out, const float* in, float b, unsigned int pitch, unsigned int width, unsigned int height, unsigned int padX, unsigned int padY)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	unsigned int y = threadIdx.y + 16*blockIdx.y;

	// Sacrifice the border pixels to simplify the implementation. 
	if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
		float* d = (float*)in;
		float* m = (float*)out;

		unsigned int o2 = (y+padY)*pitch+x+padX; // On this row.
		unsigned int o1 = o2 - pitch; // On previous row.
		unsigned int o3 = o2 + pitch; // On next row.

		m[o2] = (1.0f-b) * d[o2] + b * 0.125f * (d[o1 - 1] + d[o1] + d[o1 + 1] + d[o2 - 1] + d[o2 + 1] + d[o3 - 1] + d[o3] + d[o3 + 1]);
	}
}


void dartSmoothing(float* out, const float* in, float b, unsigned int radius, unsigned int width, unsigned int height)
{
	float* D_inData;
	float* D_outData;

	unsigned int pitch;
	allocateVolume(D_inData, width+2, height+2, pitch);
	copyVolumeToDevice(in, width, width, height, D_inData, pitch);

	allocateVolume(D_outData, width+2, height+2, pitch);
	zeroVolume(D_outData, pitch, width+2, height+2);

	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);
	if (radius == 1)
		devDartSmoothing<<<gridSize, blockSize>>>(D_outData, D_inData, b, pitch, width, height, 1, 1);
	else
		devDartSmoothingRadius<<<gridSize, blockSize>>>(D_outData, D_inData, b, radius, pitch, width, height, 1, 1);

	copyVolumeFromDevice(out, width, width, height, D_outData, pitch);

	hipFree(D_outData);
	hipFree(D_inData);

}



bool setGPUIndex(int iGPUIndex)
{
	hipSetDevice(iGPUIndex);
	hipError_t err = hipGetLastError();

	// Ignore errors caused by calling hipSetDevice multiple times
	if (err != hipSuccess && err != hipErrorSetOnActiveProcess)
		return false;

	return true;
}


}
