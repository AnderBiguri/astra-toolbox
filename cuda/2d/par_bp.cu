#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>
#include <iostream>

#include "util.h"
#include "arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

#define PIXELTRACE


typedef texture<float, 2, hipReadModeElementType> texture2D;

static texture2D gT_projTexture;


namespace astraCUDA {

const unsigned int g_anglesPerBlock = 16;
const unsigned int g_blockSliceSize = 32;
const unsigned int g_blockSlices = 16;

const unsigned int g_MaxAngles = 2560;

__constant__ float gC_angle_sin[g_MaxAngles];
__constant__ float gC_angle_cos[g_MaxAngles];
__constant__ float gC_angle_offset[g_MaxAngles];

static bool bindProjDataTexture(float* data, unsigned int pitch, unsigned int width, unsigned int height)
{
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

	gT_projTexture.addressMode[0] = hipAddressModeClamp;
	gT_projTexture.addressMode[1] = hipAddressModeClamp;
	gT_projTexture.filterMode = hipFilterModeLinear;
	gT_projTexture.normalized = false;

	hipBindTexture2D(0, gT_projTexture, (const void*)data, channelDesc, width, height, sizeof(float)*pitch);

	// TODO: error value?

	return true;
}

__global__ void devBP(float* D_volData, unsigned int volPitch, unsigned int startAngle, bool offsets, const SDimensions dims)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f ) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f ) / dims.fDetScale;

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;
	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 1.5f;

	if (offsets) {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];
			const float TOffset = gC_angle_offset[angle];

			const float fT = fT_base + fX * cos_theta - fY * sin_theta + TOffset;
			fVal += tex2D(gT_projTexture, fT, fA);
			fA += 1.0f;
		}

	} else {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];

			const float fT = fT_base + fX * cos_theta - fY * sin_theta;
			fVal += tex2D(gT_projTexture, fT, fA);
			fA += 1.0f;
		}

	}

	volData[(Y+1)*volPitch+X+1] += fVal;
}

// supersampling version
__global__ void devBP_SS(float* D_volData, unsigned int volPitch, unsigned int startAngle, bool offsets, const SDimensions dims)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	int endAngle = startAngle + g_anglesPerBlock;
	if (endAngle > dims.iProjAngles)
		endAngle = dims.iProjAngles;
	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f - 0.5f + 0.5f/dims.iRaysPerPixelDim) / dims.fDetScale;

	const float fSubStep = 1.0f/(dims.iRaysPerPixelDim * dims.fDetScale);

	float* volData = (float*)D_volData;

	float fVal = 0.0f;
	float fA = startAngle + 0.5f;
	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 1.5f;

	if (offsets) {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];
			const float TOffset = gC_angle_offset[angle];

			float fT = fT_base + fX * cos_theta - fY * sin_theta + TOffset;

			for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
				float fTy = fT;
				fT += fSubStep * cos_theta;
				for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
					fVal += tex2D(gT_projTexture, fTy, fA);
					fTy -= fSubStep * sin_theta;
				}
			}
			fA += 1.0f;
		}

	} else {

		for (int angle = startAngle; angle < endAngle; ++angle)
		{
			const float cos_theta = gC_angle_cos[angle];
			const float sin_theta = gC_angle_sin[angle];

			float fT = fT_base + fX * cos_theta - fY * sin_theta;

			for (int iSubX = 0; iSubX < dims.iRaysPerPixelDim; ++iSubX) {
				float fTy = fT;
				fT += fSubStep * cos_theta;
				for (int iSubY = 0; iSubY < dims.iRaysPerPixelDim; ++iSubY) {
					fVal += tex2D(gT_projTexture, fTy, fA);
					fTy -= fSubStep * sin_theta;
				}
			}
			fA += 1.0f;

		}

	}

	volData[(Y+1)*volPitch+X+1] += fVal / (dims.iRaysPerPixelDim * dims.iRaysPerPixelDim);
}

__global__ void devBP_SART(float* D_volData, unsigned int volPitch, float offset, float angle_sin, float angle_cos, const SDimensions dims)
{
	const int relX = threadIdx.x;
	const int relY = threadIdx.y;

	const int X = blockIdx.x * g_blockSlices + relX;
	const int Y = blockIdx.y * g_blockSliceSize + relY;

	if (X >= dims.iVolWidth || Y >= dims.iVolHeight)
		return;

	const float fX = ( X - 0.5f*dims.iVolWidth + 0.5f ) / dims.fDetScale;
	const float fY = ( Y - 0.5f*dims.iVolHeight + 0.5f ) / dims.fDetScale;

	const float fT_base = 0.5f*dims.iProjDets - 0.5f + 0.5f;

	const float fT = fT_base + fX * angle_cos - fY * angle_sin + offset;
	const float fVal = tex2D(gT_projTexture, fT, 0.5f);

	D_volData[(Y+1)*volPitch+X+1] += fVal;
}


bool BP_internal(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const float* angles, const float* TOffsets)
{
	// TODO: process angles block by block
	assert(dims.iProjAngles <= g_MaxAngles);

	float* angle_sin = new float[dims.iProjAngles];
	float* angle_cos = new float[dims.iProjAngles];

	bindProjDataTexture(D_projData, projPitch, dims.iProjDets+2, dims.iProjAngles);

	for (unsigned int i = 0; i < dims.iProjAngles; ++i) {
		angle_sin[i] = sinf(angles[i]);
		angle_cos[i] = cosf(angles[i]);
	}
	hipError_t e1 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_sin), angle_sin, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	hipError_t e2 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_cos), angle_cos, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
	assert(e1 == hipSuccess);
	assert(e2 == hipSuccess);

	if (TOffsets) {
		hipError_t e3 = hipMemcpyToSymbol(HIP_SYMBOL(gC_angle_offset), TOffsets, dims.iProjAngles*sizeof(float), 0, hipMemcpyHostToDevice);
		assert(e3 == hipSuccess);
	}

	delete[] angle_sin;
	delete[] angle_cos;

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	hipStream_t stream;
	hipStreamCreate(&stream);

	for (unsigned int i = 0; i < dims.iProjAngles; i += g_anglesPerBlock) {

		if (dims.iRaysPerPixelDim > 1)
			devBP_SS<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, (TOffsets != 0), dims);
		else
			devBP<<<dimGrid, dimBlock, 0, stream>>>(D_volumeData, volumePitch, i, (TOffsets != 0), dims);
	}
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	hipStreamDestroy(stream);

	return true;
}

bool BP(float* D_volumeData, unsigned int volumePitch,
        float* D_projData, unsigned int projPitch,
        const SDimensions& dims, const float* angles, const float* TOffsets)
{
	for (unsigned int iAngle = 0; iAngle < dims.iProjAngles; iAngle += g_MaxAngles) {
		SDimensions subdims = dims;
		unsigned int iEndAngle = iAngle + g_MaxAngles;
		if (iEndAngle >= dims.iProjAngles)
			iEndAngle = dims.iProjAngles;
		subdims.iProjAngles = iEndAngle - iAngle;

		bool ret;
		ret = BP_internal(D_volumeData, volumePitch,
		                  D_projData + iAngle * projPitch, projPitch,
		                  subdims, angles + iAngle,
		                  TOffsets ? TOffsets + iAngle : 0);
		if (!ret)
			return false;
	}
	return true;
}


bool BP_SART(float* D_volumeData, unsigned int volumePitch,
             float* D_projData, unsigned int projPitch,
             unsigned int angle, const SDimensions& dims,
             const float* angles, const float* TOffsets)
{
	// only one angle
	bindProjDataTexture(D_projData, projPitch, dims.iProjDets, 1);

	float angle_sin = sinf(angles[angle]);
	float angle_cos = cosf(angles[angle]);

	float offset = 0.0f;
	if (TOffsets)
		offset = TOffsets[angle];

	dim3 dimBlock(g_blockSlices, g_blockSliceSize);
	dim3 dimGrid((dims.iVolWidth+g_blockSlices-1)/g_blockSlices,
	             (dims.iVolHeight+g_blockSliceSize-1)/g_blockSliceSize);

	devBP_SART<<<dimGrid, dimBlock>>>(D_volumeData, volumePitch, offset, angle_sin, angle_cos, dims);
	hipDeviceSynchronize();

	cudaTextForceKernelsCompletion();

	return true;
}


}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_projData;

	SDimensions dims;
	dims.iVolWidth = 1024;
	dims.iVolHeight = 1024;
	dims.iProjAngles = 512;
	dims.iProjDets = 1536;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;

	unsigned int volumePitch, projPitch;

	allocateVolume(D_volumeData, dims.iVolWidth+2, dims.iVolHeight+2, volumePitch);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_projData, dims.iProjDets+2, dims.iProjAngles, projPitch);
	printf("pitch: %u\n", projPitch);

	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	memset(img, 0, dims.iVolWidth*dims.iVolHeight*sizeof(float));

	copyVolumeToDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);
	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_projData, projPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	BP(D_volumeData, volumePitch, D_projData, projPitch, dims, angle, 0);

	delete[] angle;

	copyVolumeFromDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif
