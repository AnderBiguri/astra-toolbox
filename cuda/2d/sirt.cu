#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include <cstdio>
#include <cassert>

#include "sirt.h"
#include "util.h"
#include "arith.h"

#ifdef STANDALONE
#include "testutil.h"
#endif

namespace astraCUDA {

SIRT::SIRT() : ReconAlgo()
{
	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;
	D_pixelWeight = 0;

	D_minMaskData = 0;
	D_maxMaskData = 0;

	freeMinMaxMasks = false;
}


SIRT::~SIRT()
{
	reset();
}

void SIRT::reset()
{
	hipFree(D_projData);
	hipFree(D_tmpData);
	hipFree(D_lineWeight);
	hipFree(D_pixelWeight);
	if (freeMinMaxMasks) {
		hipFree(D_minMaskData);
		hipFree(D_maxMaskData);
	}

	D_projData = 0;
	D_tmpData = 0;

	D_lineWeight = 0;
	D_pixelWeight = 0;

	freeMinMaxMasks = false;
	D_minMaskData = 0;
	D_maxMaskData = 0;

	useVolumeMask = false;
	useSinogramMask = false;

	ReconAlgo::reset();
}

bool SIRT::init()
{
	allocateVolume(D_pixelWeight, dims.iVolWidth+2, dims.iVolHeight+2, pixelPitch);
	zeroVolume(D_pixelWeight, pixelPitch, dims.iVolWidth+2, dims.iVolHeight+2);

	allocateVolume(D_tmpData, dims.iVolWidth+2, dims.iVolHeight+2, tmpPitch);
	zeroVolume(D_tmpData, tmpPitch, dims.iVolWidth+2, dims.iVolHeight+2);

	allocateVolume(D_projData, dims.iProjDets+2, dims.iProjAngles, projPitch);
	zeroVolume(D_projData, projPitch, dims.iProjDets+2, dims.iProjAngles);
	
	allocateVolume(D_lineWeight, dims.iProjDets+2, dims.iProjAngles, linePitch);
	zeroVolume(D_lineWeight, linePitch, dims.iProjDets+2, dims.iProjAngles);

	// We can't precompute lineWeights and pixelWeights when using a mask
	if (!useVolumeMask && !useSinogramMask)
		precomputeWeights();

	// TODO: check if allocations succeeded
	return true;
}

bool SIRT::precomputeWeights()
{
	zeroVolume(D_lineWeight, linePitch, dims.iProjDets+2, dims.iProjAngles);
	if (useVolumeMask) {
		callFP(D_maskData, maskPitch, D_lineWeight, linePitch, 1.0f);
	} else {
		processVol<opSet, VOL>(D_tmpData, 1.0f, tmpPitch, dims.iVolWidth, dims.iVolHeight);
		callFP(D_tmpData, tmpPitch, D_lineWeight, linePitch, 1.0f);
	}
	processVol<opInvert, SINO>(D_lineWeight, linePitch, dims.iProjDets, dims.iProjAngles);

	if (useSinogramMask) {
		// scale line weights with sinogram mask to zero out masked sinogram pixels
		processVol<opMul, SINO>(D_lineWeight, D_smaskData, linePitch, dims.iProjDets, dims.iProjAngles);
	}


	zeroVolume(D_pixelWeight, pixelPitch, dims.iVolWidth+2, dims.iVolHeight+2);
	if (useSinogramMask) {
		callBP(D_pixelWeight, pixelPitch, D_smaskData, smaskPitch);
	} else {
		processVol<opSet, SINO>(D_projData, 1.0f, projPitch, dims.iProjDets, dims.iProjAngles);
		callBP(D_pixelWeight, pixelPitch, D_projData, projPitch);
	}
	processVol<opInvert, VOL>(D_pixelWeight, pixelPitch, dims.iVolWidth, dims.iVolHeight);

	if (useVolumeMask) {
		// scale pixel weights with mask to zero out masked pixels
		processVol<opMul, VOL>(D_pixelWeight, D_maskData, pixelPitch, dims.iVolWidth, dims.iVolHeight);
	}

	return true;
}

bool SIRT::setMinMaxMasks(float* D_minMaskData_, float* D_maxMaskData_,
	                      unsigned int iPitch)
{
	D_minMaskData = D_minMaskData_;
	D_maxMaskData = D_maxMaskData_;
	minMaskPitch = iPitch;
	maxMaskPitch = iPitch;

	freeMinMaxMasks = false;
	return true;
}

bool SIRT::uploadMinMaxMasks(const float* pfMinMaskData, const float* pfMaxMaskData,
	                         unsigned int iPitch)
{
	freeMinMaxMasks = true;
	bool ok = true;
	if (pfMinMaskData) {
		allocateVolume(D_minMaskData, dims.iVolWidth+2, dims.iVolHeight+2, minMaskPitch);
		ok = copyVolumeToDevice(pfMinMaskData, iPitch,
		                        dims.iVolWidth, dims.iVolHeight,
		                        D_minMaskData, minMaskPitch);
	}
	if (!ok)
		return false;

	if (pfMaxMaskData) {
		allocateVolume(D_maxMaskData, dims.iVolWidth+2, dims.iVolHeight+2, maxMaskPitch);
		ok = copyVolumeToDevice(pfMaxMaskData, iPitch,
		                        dims.iVolWidth, dims.iVolHeight,
		                        D_maxMaskData, maxMaskPitch);
	}
	if (!ok)
		return false;

	return true;
}

bool SIRT::iterate(unsigned int iterations)
{
	shouldAbort = false;

	if (useVolumeMask || useSinogramMask)
		precomputeWeights();

	// iteration
	for (unsigned int iter = 0; iter < iterations && !shouldAbort; ++iter) {

		// copy sinogram to projection data
		hipMemcpy2D(D_projData, sizeof(float)*projPitch, D_sinoData, sizeof(float)*sinoPitch, sizeof(float)*(dims.iProjDets+2), dims.iProjAngles, hipMemcpyDeviceToDevice);

		// do FP, subtracting projection from sinogram
		if (useVolumeMask) {
				hipMemcpy2D(D_tmpData, sizeof(float)*tmpPitch, D_volumeData, sizeof(float)*volumePitch, sizeof(float)*(dims.iVolWidth+2), dims.iVolHeight+2, hipMemcpyDeviceToDevice);
				processVol<opMul, VOL>(D_tmpData, D_maskData, tmpPitch, dims.iVolWidth, dims.iVolHeight);
				callFP(D_tmpData, tmpPitch, D_projData, projPitch, -1.0f);
		} else {
				callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);
		}

		processVol<opMul, SINO>(D_projData, D_lineWeight, projPitch, dims.iProjDets, dims.iProjAngles);

		zeroVolume(D_tmpData, tmpPitch, dims.iVolWidth+2, dims.iVolHeight+2);

		callBP(D_tmpData, tmpPitch, D_projData, projPitch);

		processVol<opAddMul, VOL>(D_volumeData, D_pixelWeight, D_tmpData, volumePitch, dims.iVolWidth, dims.iVolHeight);

		if (useMinConstraint)
			processVol<opClampMin, VOL>(D_volumeData, fMinConstraint, volumePitch, dims.iVolWidth, dims.iVolHeight);
		if (useMaxConstraint)
			processVol<opClampMax, VOL>(D_volumeData, fMaxConstraint, volumePitch, dims.iVolWidth, dims.iVolHeight);
		if (D_minMaskData)
			processVol<opClampMinMask, VOL>(D_volumeData, D_minMaskData, volumePitch, dims.iVolWidth, dims.iVolHeight);
		if (D_maxMaskData)
			processVol<opClampMaxMask, VOL>(D_volumeData, D_maxMaskData, volumePitch, dims.iVolWidth, dims.iVolHeight);
	}

	return true;
}

float SIRT::computeDiffNorm()
{
	// copy sinogram to projection data
	hipMemcpy2D(D_projData, sizeof(float)*projPitch, D_sinoData, sizeof(float)*sinoPitch, sizeof(float)*(dims.iProjDets+2), dims.iProjAngles, hipMemcpyDeviceToDevice);

	// do FP, subtracting projection from sinogram
	if (useVolumeMask) {
			hipMemcpy2D(D_tmpData, sizeof(float)*tmpPitch, D_volumeData, sizeof(float)*volumePitch, sizeof(float)*(dims.iVolWidth+2), dims.iVolHeight+2, hipMemcpyDeviceToDevice);
			processVol<opMul, VOL>(D_tmpData, D_maskData, tmpPitch, dims.iVolWidth, dims.iVolHeight);
			callFP(D_tmpData, tmpPitch, D_projData, projPitch, -1.0f);
	} else {
			callFP(D_volumeData, volumePitch, D_projData, projPitch, -1.0f);
	}


	// compute norm of D_projData

	float s = dotProduct2D(D_projData, projPitch, dims.iProjDets, dims.iProjAngles, 1, 0);

	return sqrt(s);
}


bool doSIRT(float* D_volumeData, unsigned int volumePitch,
            float* D_sinoData, unsigned int sinoPitch,
            float* D_maskData, unsigned int maskPitch,
            const SDimensions& dims, const float* angles,
            const float* TOffsets, unsigned int iterations)
{
	SIRT sirt;
	bool ok = true;

	ok &= sirt.setGeometry(dims, angles);
	if (D_maskData)
		ok &= sirt.enableVolumeMask();
	if (TOffsets)
		ok &= sirt.setTOffsets(TOffsets);

	if (!ok)
		return false;

	ok = sirt.init();
	if (!ok)
		return false;

	if (D_maskData)
		ok &= sirt.setVolumeMask(D_maskData, maskPitch);

	ok &= sirt.setBuffers(D_volumeData, volumePitch, D_sinoData, sinoPitch);
	if (!ok)
		return false;

	ok = sirt.iterate(iterations);

	return ok;
}

}

#ifdef STANDALONE

using namespace astraCUDA;

int main()
{
	float* D_volumeData;
	float* D_sinoData;

	SDimensions dims;
	dims.iVolWidth = 1024;
	dims.iVolHeight = 1024;
	dims.iProjAngles = 512;
	dims.iProjDets = 1536;
	dims.fDetScale = 1.0f;
	dims.iRaysPerDet = 1;
	unsigned int volumePitch, sinoPitch;

	allocateVolume(D_volumeData, dims.iVolWidth+2, dims.iVolHeight+2, volumePitch);
	zeroVolume(D_volumeData, volumePitch, dims.iVolWidth+2, dims.iVolHeight+2);
	printf("pitch: %u\n", volumePitch);

	allocateVolume(D_sinoData, dims.iProjDets+2, dims.iProjAngles, sinoPitch);
	zeroVolume(D_sinoData, sinoPitch, dims.iProjDets+2, dims.iProjAngles);
	printf("pitch: %u\n", sinoPitch);
	
	unsigned int y, x;
	float* sino = loadImage("sino.png", y, x);

	float* img = new float[dims.iVolWidth*dims.iVolHeight];

	copySinogramToDevice(sino, dims.iProjDets, dims.iProjDets, dims.iProjAngles, D_sinoData, sinoPitch);

	float* angle = new float[dims.iProjAngles];

	for (unsigned int i = 0; i < dims.iProjAngles; ++i)
		angle[i] = i*(M_PI/dims.iProjAngles);

	SIRT sirt;

	sirt.setGeometry(dims, angle);
	sirt.init();

	sirt.setBuffers(D_volumeData, volumePitch, D_sinoData, sinoPitch);

	sirt.iterate(25);


	delete[] angle;

	copyVolumeFromDevice(img, dims.iVolWidth, dims.iVolWidth, dims.iVolHeight, D_volumeData, volumePitch);

	saveImage("vol.png",dims.iVolHeight,dims.iVolWidth,img);

	return 0;
}
#endif

