/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include "util.h"
#include "dataop.h"
#include "arith.h"
#include <cassert>

namespace astraCUDA {

void operationVolumeMult(float* data1, float* data2, unsigned int width, unsigned int height)
{
	float* D_data1;
	float* D_data2;

	unsigned int pitch;
	allocateVolume(D_data1, width, height, pitch);
	copyVolumeToDevice(data1, width, width, height, D_data1, pitch);

	allocateVolume(D_data2, width, height, pitch);
	copyVolumeToDevice(data2, width, width, height, D_data2, pitch);

	processVol<opMul, VOL>(D_data1, D_data2, pitch, width, height);

	copyVolumeFromDevice(data1, width, width, height, D_data1, pitch);

	hipFree(D_data1);
	hipFree(D_data2);
}

void operationVolumeMultScalarMask(float* data, float* mask, float scalar, unsigned int width, unsigned int height)
{
	float* D_data;
	float* D_mask;

	unsigned int pitch;
	allocateVolume(D_data, width, height, pitch);
	copyVolumeToDevice(data, width, width, height, D_data, pitch);

	allocateVolume(D_mask, width, height, pitch);
	copyVolumeToDevice(mask, width, width, height, D_mask, pitch);

	processVol<opMulMask, VOL>(D_data, D_mask, scalar, pitch, width, height);

	copyVolumeFromDevice(data, width, width, height, D_data, pitch);

	hipFree(D_data);
	hipFree(D_mask);
}


void operationVolumeMultScalar(float* data, float scalar, unsigned int width, unsigned int height)
{
	float* D_data;

	unsigned int pitch;
	allocateVolume(D_data, width, height, pitch);
	copyVolumeToDevice(data, width, width, height, D_data, pitch);

	processVol<opMul, VOL>(D_data, scalar, pitch, width, height);

	copyVolumeFromDevice(data, width, width, height, D_data, pitch);

	hipFree(D_data);
}


void operationVolumeAdd(float* data1, float* data2, unsigned int width, unsigned int height)
{
	float* D_data1;
	float* D_data2;

	unsigned int pitch;
	allocateVolume(D_data1, width, height, pitch);
	copyVolumeToDevice(data1, width, width, height, D_data1, pitch);

	allocateVolume(D_data2, width, height, pitch);
	copyVolumeToDevice(data2, width, width, height, D_data2, pitch);

	processVol<opAdd, VOL>(D_data1, D_data2, pitch, width, height);

	copyVolumeFromDevice(data1, width, width, height, D_data1, pitch);

	hipFree(D_data1);
	hipFree(D_data2);
}


void operationVolumeAddScalar(float* data, float scalar, unsigned int width, unsigned int height)
{
	float* D_data;

	unsigned int pitch;
	allocateVolume(D_data, width, height, pitch);
	copyVolumeToDevice(data, width, width, height, D_data, pitch);

	processVol<opAdd, VOL>(D_data, scalar, pitch, width, height);

	copyVolumeFromDevice(data, width, width, height, D_data, pitch);

	hipFree(D_data);
}


}
