#include "hip/hip_runtime.h"
/*
-----------------------------------------------------------------------
Copyright 2012 iMinds-Vision Lab, University of Antwerp

Contact: astra@ua.ac.be
Website: http://astra.ua.ac.be


This file is part of the
All Scale Tomographic Reconstruction Antwerp Toolbox ("ASTRA Toolbox").

The ASTRA Toolbox is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

The ASTRA Toolbox is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with the ASTRA Toolbox. If not, see <http://www.gnu.org/licenses/>.

-----------------------------------------------------------------------
$Id$
*/

#include "util.h"
#include "arith.h"
#include <cassert>

namespace astraCUDA {


struct opAddScaled {
	__device__ void operator()(float& out, const float in, const float inp) {
		out += in * inp;
	}
};
struct opScaleAndAdd {
	__device__ void operator()(float& out, const float in, const float inp) {
		out = in + out * inp;
	}
};
struct opAddMulScaled {
	__device__ void operator()(float& out, const float in1, const float in2, const float inp) {
		out += in1 * in2 * inp;
	}
};
struct opAddMul {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 * in2;
	}
};
struct opAdd {
	__device__ void operator()(float& out, const float in) {
		out += in;
	}
};
struct opAdd2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out += in1 + in2;
	}
};
struct opMul {
	__device__ void operator()(float& out, const float in) {
		out *= in;
	}
};
struct opMul2 {
	__device__ void operator()(float& out, const float in1, const float in2) {
		out *= in1 * in2;
	}
};
struct opDividedBy {
	__device__ void operator()(float& out, const float in) {
		if (out > 0.000001f) // out is assumed to be positive
			out = in / out;
		else
			out = 0.0f;
	}
};
struct opInvert {
	__device__ void operator()(float& out) {
		if (out > 0.000001f) // out is assumed to be positive
			out = 1 / out;
		else
			out = 0.0f;
	}
};
struct opSet {
	__device__ void operator()(float& out, const float inp) {
		out = inp;
	}
};
struct opClampMin {
	__device__ void operator()(float& out, const float inp) {
		if (out < inp)
			out = inp;
	}
};
struct opClampMax {
	__device__ void operator()(float& out, const float inp) {
		if (out > inp)
			out = inp;
	}
};
struct opClampMinMask {
	__device__ void operator()(float& out, const float in) {
		if (out < in)
			out = in;
	}
};
struct opClampMaxMask {
	__device__ void operator()(float& out, const float in) {
		if (out > in)
			out = in;
	}
};
struct opSetMaskedValues {
	__device__ void operator()(float& out, const float in, const float inp) {
		if (!in)
			out = inp;
	}
};
struct opSegmentAndMask {
	__device__ void operator()(float& out1, float& out2, const float inp1, const float inp2) {
		if (out1 >= inp1) {
			out1 = inp2;
			out2 = 0.0f;
		}

	}

};
struct opMulMask {
	__device__ void operator()(float& out, const float mask, const float in) {
		if (mask > 0.0f) {
			out *= in;
		}
	}
};



template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devtoD(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devFtoD(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devFFtoDD(float* pfOut1, float* pfOut2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut1[off], pfOut2[off], fParam1, fParam2);
		off += pitch;
		y++;
	}
}



template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devDtoD(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devDFtoD(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn[off], fParam);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devDDtoD(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off]);
		off += pitch;
		y++;
	}
}

template<class op, unsigned int padX, unsigned int padY, unsigned int repeat>
__global__ void devDDFtoD(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	unsigned int x = threadIdx.x + 16*blockIdx.x;
	if (x >= width) return;

	unsigned int y = (threadIdx.y + 16*blockIdx.y)*repeat;
	unsigned int off = (y+padY)*pitch+x+padX;
	for (unsigned int i = 0; i < repeat; ++i) {
		if (y >= height)
			break;
		op()(pfOut[off], pfIn1[off], pfIn2[off], fParam);
		off += pitch;
		y++;
	}
}
















template<typename op, VolType t>
void processVolCopy(float* out, unsigned int width, unsigned int height)
{
	float* D_out;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);

	processVol<op, t>(D_out, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
}

template<typename op, VolType t>
void processVolCopy(float* out, float param, unsigned int width, unsigned int height)
{
	float* D_out;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);

	processVol<op, t>(D_out, param, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
}

template<typename op, VolType t>
void processVolCopy(float* out1, float* out2, float param1, float param2, unsigned int width, unsigned int height)
{
	float* D_out1;
	float* D_out2;

	unsigned int pitch;
	allocateVolume(D_out1, width+2, height+2, pitch);
	copyVolumeToDevice(out1, width, width, height, D_out1, pitch);
	allocateVolume(D_out2, width+2, height+2, pitch);
	copyVolumeToDevice(out2, width, width, height, D_out2, pitch);

	processVol<op, t>(D_out1, D_out2, param1, param2, pitch, width, height);

	copyVolumeFromDevice(out1, width, width, height, D_out1, pitch);
	copyVolumeFromDevice(out2, width, width, height, D_out2, pitch);

	hipFree(D_out1);
	hipFree(D_out2);
}


template<typename op, VolType t>
void processVolCopy(float* out, const float* in, unsigned int width, unsigned int height)
{
	float* D_out;
	float* D_in;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);
	allocateVolume(D_in, width+2, height+2, pitch);
	copyVolumeToDevice(in, width, width, height, D_in, pitch);

	processVol<op, t>(D_out, D_in, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in);
}

template<typename op, VolType t>
void processVolCopy(float* out, const float* in, float param, unsigned int width, unsigned int height)
{
	float* D_out;
	float* D_in;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);
	allocateVolume(D_in, width+2, height+2, pitch);
	copyVolumeToDevice(in, width, width, height, D_in, pitch);

	processVol<op, t>(D_out, D_in, param, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in);
}

template<typename op, VolType t>
void processVolCopy(float* out, const float* in1, const float* in2, unsigned int width, unsigned int height)
{
	float* D_out;
	float* D_in1;
	float* D_in2;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);
	allocateVolume(D_in1, width+2, height+2, pitch);
	copyVolumeToDevice(in1, width, width, height, D_in1, pitch);
	allocateVolume(D_in2, width+2, height+2, pitch);
	copyVolumeToDevice(in2, width, width, height, D_in2, pitch);

	processVol<op, t>(D_out, D_in1, D_in2, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in1);
	hipFree(D_in2);
}

template<typename op, VolType t>
void processVolCopy(float* out, const float* in1, const float* in2, float param, unsigned int width, unsigned int height)
{
	float* D_out;
	float* D_in1;
	float* D_in2;

	unsigned int pitch;
	allocateVolume(D_out, width+2, height+2, pitch);
	copyVolumeToDevice(out, width, width, height, D_out, pitch);
	allocateVolume(D_in1, width+2, height+2, pitch);
	copyVolumeToDevice(in1, width, width, height, D_in1, pitch);
	allocateVolume(D_in2, width+2, height+2, pitch);
	copyVolumeToDevice(in2, width, width, height, D_in2, pitch);

	processVol<op, t>(D_out, D_in1, D_in2, param, pitch, width, height);

	copyVolumeFromDevice(out, width, width, height, D_out, pitch);

	hipFree(D_out);
	hipFree(D_in1);
	hipFree(D_in2);
}









template<typename op, VolType t>
void processVol(float* pfOut, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+511)/512);

	devtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

template<typename op, VolType t>
void processVol(float* pfOut, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devFtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, fParam, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

template<typename op, VolType t>
void processVol(float* pfOut1, float* pfOut2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devFFtoDD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut1, pfOut2, fParam1, fParam2, pitch, width, height);

	cudaTextForceKernelsCompletion();
}


template<typename op, VolType t>
void processVol(float* pfOut, const float* pfIn, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, pfIn, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

template<typename op, VolType t>
void processVol(float* pfOut, const float* pfIn, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDFtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

template<typename op, VolType t>
void processVol(float* pfOut, const float* pfIn1, const float* pfIn2, float fParam, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDDFtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

template<typename op, VolType t>
void processVol(float* pfOut, const float* pfIn1, const float* pfIn2, unsigned int pitch, unsigned int width, unsigned int height)
{
	dim3 blockSize(16,16);
	dim3 gridSize((width+15)/16, (height+15)/16);

	devDDtoD<op, 1, t, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, pitch, width, height);

	cudaTextForceKernelsCompletion();
}

















template<typename op>
void processVol3D(hipPitchedPtr& out, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processVol3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processVol3D(hipPitchedPtr& out1, hipPitchedPtr& out2, float fParam1, float fParam2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut1 = (float*)out1.ptr;
	float *pfOut2 = (float*)out2.ptr;
	unsigned int step = out1.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devFFtoDD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut1, pfOut2, fParam1, fParam2, out1.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut1 += step;
		pfOut2 += step;
	}

	cudaTextForceKernelsCompletion();
}


template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processVol3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iVolX+15)/16, (dims.iVolY+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iVolY;

	for (unsigned int i = 0; i < dims.iVolZ; ++i) {
		devDDtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iVolX, dims.iVolY);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	cudaTextForceKernelsCompletion();
}













template<typename op>
void processSino3D(hipPitchedPtr& out, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processSino3D(hipPitchedPtr& out, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processSino3D(hipPitchedPtr& out1, hipPitchedPtr& out2, float fParam1, float fParam2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut1 = (float*)out1.ptr;
	float *pfOut2 = (float*)out2.ptr;
	unsigned int step = out1.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devFFtoDD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut1, pfOut2, fParam1, fParam2, out1.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut1 += step;
		pfOut2 += step;
	}

	cudaTextForceKernelsCompletion();
}


template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn = (float*)in.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDFtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, fParam, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	cudaTextForceKernelsCompletion();
}

template<typename op>
void processSino3D(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims)
{
	dim3 blockSize(16,16);
	dim3 gridSize((dims.iProjU+15)/16, (dims.iProjAngles+511)/512);
	float *pfOut = (float*)out.ptr;
	float *pfIn1 = (float*)in1.ptr;
	float *pfIn2 = (float*)in2.ptr;
	unsigned int step = out.pitch/sizeof(float) * dims.iProjAngles;

	for (unsigned int i = 0; i < dims.iProjV; ++i) {
		devDDtoD<op, 0, 0, 32><<<gridSize, blockSize>>>(pfOut, pfIn1, pfIn2, out.pitch/sizeof(float), dims.iProjU, dims.iProjAngles);
		pfOut += step;
		pfIn1 += step;
		pfIn2 += step;
	}

	cudaTextForceKernelsCompletion();
}


















#define INST_DFtoD(name) \
  template void processVolCopy<name, VOL>(float* out, const float* in, float param, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, const float* in, float param, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, const float* in, float param, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, const float* in, float param, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, float fParam, const SDimensions3D& dims);

#define INST_DtoD(name) \
  template void processVolCopy<name, VOL>(float* out, const float* in, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, const float* in, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, const float* in, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, const float* in, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in, const SDimensions3D& dims);

#define INST_DDtoD(name) \
  template void processVolCopy<name, VOL>(float* out, const float* in1, const float* in2, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, const float* in1, const float* in2, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, const float* in1, const float* in2, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, const float* in1, const float* in2, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, const SDimensions3D& dims);

#define INST_DDFtoD(name) \
  template void processVolCopy<name, VOL>(float* out, const float* in1, const float* in2, float fParam, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, const float* in1, const float* in2, float fParam, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, const float* in1, const float* in2, float fParam, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, const float* in1, const float* in2, float fParam, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const hipPitchedPtr& in1, const hipPitchedPtr& in2, float fParam, const SDimensions3D& dims);


#define INST_toD(name) \
  template void processVolCopy<name, VOL>(float* out, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, const SDimensions3D& dims);

#define INST_FtoD(name) \
  template void processVolCopy<name, VOL>(float* out, float param, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out, float param, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out, float param, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out, float param, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out, float param, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out, float param, const SDimensions3D& dims);

#define INST_FFtoDD(name) \
  template void processVolCopy<name, VOL>(float* out1, float* out2, float fParam1, float fParam2, unsigned int width, unsigned int height); \
  template void processVolCopy<name, SINO>(float* out1, float* out2, float fParam1, float fParam2, unsigned int width, unsigned int height); \
  template void processVol<name, VOL>(float* out1, float* out2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol<name, SINO>(float* out1, float* out2, float fParam1, float fParam2, unsigned int pitch, unsigned int width, unsigned int height); \
  template void processVol3D<name>(hipPitchedPtr& out1, hipPitchedPtr& out2, float fParam1, float fParam2, const SDimensions3D& dims); \
  template void processSino3D<name>(hipPitchedPtr& out1, hipPitchedPtr& out2, float fParam1, float fParam2, const SDimensions3D& dims);



INST_DFtoD(opAddScaled)
INST_DFtoD(opScaleAndAdd)
INST_DDFtoD(opAddMulScaled)
INST_DDtoD(opAddMul)
INST_DDtoD(opMul2)
INST_DDtoD(opAdd2)
INST_DtoD(opMul)
INST_DDtoD(opMulMask)
INST_DtoD(opAdd)
INST_DtoD(opDividedBy)
INST_toD(opInvert)
INST_FtoD(opSet)
INST_FtoD(opMul)
INST_DFtoD(opMulMask)
INST_FtoD(opAdd)
INST_FtoD(opClampMin)
INST_FtoD(opClampMax)
INST_DtoD(opClampMinMask)
INST_DtoD(opClampMaxMask)

// PDART-specific:
INST_DFtoD(opSetMaskedValues)
INST_FFtoDD(opSegmentAndMask)

}
